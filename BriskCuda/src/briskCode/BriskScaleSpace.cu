#include "hip/hip_runtime.h"

/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */


#include "BriskScaleSpace.cuh"


// construct the image pyramids
void
BriskScaleSpace::constructPyramid(const PtrStepSzb& image)
{
  assert( layers_ == 8 );

  const int octaves2 = layers_;

  pyramid_[0] = BriskLayerOne(image);
  pyramid_[1] = BriskLayerOne(pyramid_[0], BriskLayerOne::CommonParams::TWOTHIRDSAMPLE);

  for (int i = 2; i < octaves2; i += 2)
  {
    pyramid_[i] = BriskLayerOne(BriskLayerOne(pyramid_[i - 2], BriskLayerOne::CommonParams::HALFSAMPLE));
    pyramid_[i+1] = BriskLayerOne(BriskLayerOne(pyramid_[i - 1], BriskLayerOne::CommonParams::HALFSAMPLE));
  }
}


/***
 * todo: 加速
 * @param threshold_
 * @param keypoints
 */
void
BriskScaleSpace::getKeypoints(const int threshold_, float2* keypoints, float* kpSize, float* kpScore)
{

  int maxLayersPoints = 0;
  // assign thresholds
  int safeThreshold_ = (int)(threshold_ * safetyFactor_);
 // std::vector<std::vector<cv::KeyPoint> > agastPoints;

  float* scoreTemp;
  newArray( scoreTemp, maxPointNow, false   );

  //agastPoints.resize(layers_);

  // go through the octaves and intra layers and calculate agast corner scores:
  for (int i = 0; i < layers_; i++)
  {
	  newArray( kpsLoc[i], maxPointNow, false   );
    // call OAST16_9 without nms
	  BriskLayerOne& l = pyramid_[i];
	  kpsCount[i] = l.getAgastPoints(safeThreshold_, kpsLoc[i],scoreTemp); //todo: 并行化
	  maxLayersPoints = kpsCount[i] > maxLayersPoints? kpsCount[i]: maxLayersPoints;
  }

  if (layers_ == 1)
  {

	//todo: need a global kernel,optmize kernal gird and block
    // just do a simple 2d subpixel refinement...
    //const size_t num = agastPoints[0].size();

	void* counter_ptr;
	hipGetSymbolAddress(&counter_ptr, HIP_SYMBOL(g_counter)) ;

	hipMemsetAsync(counter_ptr, 0, sizeof(unsigned int));

    refineKernel1<<<kpsCount[0]/(32*4)+1,32*4,0>>>(  *this,  keypoints,  kpSize,  kpScore, 0 );


	hipGetLastError() ;//todo: cudaSafeCall

	hipMemcpyAsync(&kpsCountAfter[0], counter_ptr, sizeof(unsigned int), hipMemcpyDeviceToHost) ;//todo: cudaSafeCall

	hipStreamSynchronize(NULL) ;//todo: cudaSafeCall

   /* for (size_t n = 0; n < num; n++)
    {
      const cv::Point2f& point = agastPoints.at(0)[n].pt;
      // first check if it is a maximum:
      //非极大值抑制
      if (!isMax2D(0, (int)point.x, (int)point.y))
        continue;

      // let's do the subpixel and float scale refinement:
      BriskLayerOne& l = pyramid_[0];
      int s_0_0 = l.getAgastScore(point.x - 1, point.y - 1, 1);
      int s_1_0 = l.getAgastScore(point.x, point.y - 1, 1);
      int s_2_0 = l.getAgastScore(point.x + 1, point.y - 1, 1);
      int s_2_1 = l.getAgastScore(point.x + 1, point.y, 1);
      int s_1_1 = l.getAgastScore(point.x, point.y, 1);
      int s_0_1 = l.getAgastScore(point.x - 1, point.y, 1);
      int s_0_2 = l.getAgastScore(point.x - 1, point.y + 1, 1);
      int s_1_2 = l.getAgastScore(point.x, point.y + 1, 1);
      int s_2_2 = l.getAgastScore(point.x + 1, point.y + 1, 1);
      float delta_x, delta_y;
      float max = subpixel2D(s_0_0, s_0_1, s_0_2, s_1_0, s_1_1, s_1_2, s_2_0, s_2_1, s_2_2, delta_x, delta_y);

      // store:
      keypoints.push_back(cv::KeyPoint(float(point.x) + delta_x, float(point.y) + delta_y, basicSize_, -1, max, 0));

    }*/

    return;
  }

  float x, y, scale, score;

	void* counter_ptr;
	hipGetSymbolAddress(&counter_ptr, HIP_SYMBOL(g_counter)) ;

    hipMemsetAsync(counter_ptr, 0, sizeof(unsigned int));

    dim3 grid;
    grid.x = layers_;
    grid.y = maxLayersPoints/32;//todo optimize
    //maxLayersPoints

    refineKernel2<<<grid,32,0>>>(  *this,  keypoints,  kpSize,  kpScore, 0 );


	hipGetLastError() ;//todo: cudaSafeCall

	hipMemcpyAsync(&kpsCountAfter[0], counter_ptr, sizeof(unsigned int), hipMemcpyDeviceToHost) ;//todo: cudaSafeCall

	hipStreamSynchronize(NULL) ;//todo: cudaSafeCall

/*  for (int i = 0; i < layers_; i++)
  {
    BriskLayer& l = pyramid_[i];
    const size_t num = agastPoints[i].size();
    if (i == layers_ - 1)
    {
      for (size_t n = 0; n < num; n++)
      {
        const cv::Point2f& point = agastPoints.at(i)[n].pt;
        // consider only 2D maxima...
        if (!isMax2D(i, (int)point.x, (int)point.y))
          continue;

        bool ismax;
        float dx, dy;
        getScoreMaxBelow(i, (int)point.x, (int)point.y, l.getAgastScore(point.x, point.y, safeThreshold_), ismax, dx, dy);
        if (!ismax)
          continue;

        // get the patch on this layer:
        int s_0_0 = l.getAgastScore(point.x - 1, point.y - 1, 1);
        int s_1_0 = l.getAgastScore(point.x, point.y - 1, 1);
        int s_2_0 = l.getAgastScore(point.x + 1, point.y - 1, 1);
        int s_2_1 = l.getAgastScore(point.x + 1, point.y, 1);
        int s_1_1 = l.getAgastScore(point.x, point.y, 1);
        int s_0_1 = l.getAgastScore(point.x - 1, point.y, 1);
        int s_0_2 = l.getAgastScore(point.x - 1, point.y + 1, 1);
        int s_1_2 = l.getAgastScore(point.x, point.y + 1, 1);
        int s_2_2 = l.getAgastScore(point.x + 1, point.y + 1, 1);
        float delta_x, delta_y;
        float max = subpixel2D(s_0_0, s_0_1, s_0_2, s_1_0, s_1_1, s_1_2, s_2_0, s_2_1, s_2_2, delta_x, delta_y);

        // store:
        keypoints.push_back(
            cv::KeyPoint((float(point.x) + delta_x) * l.scale() + l.offset(),
                         (float(point.y) + delta_y) * l.scale() + l.offset(), basicSize_ * l.scale(), -1, max, i));
      }
    }
    else
    {
      // not the last layer:
      for (size_t n = 0; n < num; n++)
      {
        const cv::Point2f& point = agastPoints.at(i)[n].pt;

        // first check if it is a maximum:
        if (!isMax2D(i, (int)point.x, (int)point.y))
          continue;

        // let's do the subpixel and float scale refinement:
        bool ismax=false;

        //可见refine3D是真正判断是否最大的货色
        score = refine3D(i, (int)point.x, (int)point.y, x, y, scale, ismax);
        if (!ismax)
        {
          continue;
        }


        //理解这个basicSize的真实含义
        // finally store the detected keypoint:
        if (score > float(threshold_))
        {
          keypoints.push_back(cv::KeyPoint(x, y, basicSize_ * scale, -1, score, i));
        }
      }
    }
  }*/
}




//直接移植
// interpolated score access with recalculation when needed:
__device__ inline int
BriskScaleSpace::getScoreAbove(BriskLayerOne* layers,const int layer, const int x_layer, const int y_layer) const
{
  assert(layer < layers_-1);
  const BriskLayerOne& l = pyramid_[layer + 1];
  if (layer % 2 == 0)
  { // octave
    const int sixths_x = 4 * x_layer - 1;
    const int x_above = sixths_x / 6;
    const int sixths_y = 4 * y_layer - 1;
    const int y_above = sixths_y / 6;
    const int r_x = (sixths_x % 6);
    const int r_x_1 = 6 - r_x;
    const int r_y = (sixths_y % 6);
    const int r_y_1 = 6 - r_y;
    unsigned char score = 0xFF
        & ((r_x_1 * r_y_1 * l.getAgastScore(x_above, y_above, 1) + r_x * r_y_1
                                                                   * l.getAgastScore(x_above + 1, y_above, 1)
            + r_x_1 * r_y * l.getAgastScore(x_above, y_above + 1, 1)
            + r_x * r_y * l.getAgastScore(x_above + 1, y_above + 1, 1) + 18)
           / 36);

    return score;
  }
  else
  { // intra
    const int eighths_x = 6 * x_layer - 1;
    const int x_above = eighths_x / 8;
    const int eighths_y = 6 * y_layer - 1;
    const int y_above = eighths_y / 8;
    const int r_x = (eighths_x % 8);
    const int r_x_1 = 8 - r_x;
    const int r_y = (eighths_y % 8);
    const int r_y_1 = 8 - r_y;
    unsigned char score = 0xFF
        & ((r_x_1 * r_y_1 * l.getAgastScore(x_above, y_above, 1) + r_x * r_y_1
                                                                   * l.getAgastScore(x_above + 1, y_above, 1)
            + r_x_1 * r_y * l.getAgastScore(x_above, y_above + 1, 1)
            + r_x * r_y * l.getAgastScore(x_above + 1, y_above + 1, 1) + 32)
           / 64);
    return score;
  }
}


//直接移植
__device__ inline int
BriskScaleSpace::getScoreBelow(BriskLayerOne* layers,const int layer, const int x_layer, const int y_layer) const
{
  assert(layer);
  const BriskLayerOne& l = layers[layer - 1];
  int sixth_x;
  int quarter_x;
  float xf;
  int sixth_y;
  int quarter_y;
  float yf;

  // scaling:
  float offs;
  float area;
  int scaling;
  int scaling2;

  if (layer % 2 == 0)
  { // octave
    sixth_x = 8 * x_layer + 1;
    xf = float(sixth_x) / 6.0f;
    sixth_y = 8 * y_layer + 1;
    yf = float(sixth_y) / 6.0f;

    // scaling:
    offs = 2.0f / 3.0f;
    area = 4.0f * offs * offs;
    scaling = (int)(4194304.0 / area);
    scaling2 = (int)(float(scaling) * area);
  }
  else
  {
    quarter_x = 6 * x_layer + 1;
    xf = float(quarter_x) / 4.0f;
    quarter_y = 6 * y_layer + 1;
    yf = float(quarter_y) / 4.0f;

    // scaling:
    offs = 3.0f / 4.0f;
    area = 4.0f * offs * offs;
    scaling = (int)(4194304.0 / area);
    scaling2 = (int)(float(scaling) * area);
  }

  // calculate borders
  const float x_1 = xf - offs;
  const float x1 = xf + offs;
  const float y_1 = yf - offs;
  const float y1 = yf + offs;

  const int x_left = int(x_1 + 0.5);
  const int y_top = int(y_1 + 0.5);
  const int x_right = int(x1 + 0.5);
  const int y_bottom = int(y1 + 0.5);

  // overlap area - multiplication factors:
  const float r_x_1 = float(x_left) - x_1 + 0.5f;
  const float r_y_1 = float(y_top) - y_1 + 0.5f;
  const float r_x1 = x1 - float(x_right) + 0.5f;
  const float r_y1 = y1 - float(y_bottom) + 0.5f;
  const int dx = x_right - x_left - 1;
  const int dy = y_bottom - y_top - 1;
  const int A = (int)((r_x_1 * r_y_1) * scaling);
  const int B = (int)((r_x1 * r_y_1) * scaling);
  const int C = (int)((r_x1 * r_y1) * scaling);
  const int D = (int)((r_x_1 * r_y1) * scaling);
  const int r_x_1_i = (int)(r_x_1 * scaling);
  const int r_y_1_i = (int)(r_y_1 * scaling);
  const int r_x1_i = (int)(r_x1 * scaling);
  const int r_y1_i = (int)(r_y1 * scaling);

  // first row:
  int ret_val = A * int(l.getAgastScore(x_left, y_top, 1));
  for (int X = 1; X <= dx; X++)
  {
    ret_val += r_y_1_i * int(l.getAgastScore(x_left + X, y_top, 1));
  }
  ret_val += B * int(l.getAgastScore(x_left + dx + 1, y_top, 1));
  // middle ones:
  for (int Y = 1; Y <= dy; Y++)
  {
    ret_val += r_x_1_i * int(l.getAgastScore(x_left, y_top + Y, 1));

    for (int X = 1; X <= dx; X++)
    {
      ret_val += int(l.getAgastScore(x_left + X, y_top + Y, 1)) * scaling;
    }
    ret_val += r_x1_i * int(l.getAgastScore(x_left + dx + 1, y_top + Y, 1));
  }
  // last row:
  ret_val += D * int(l.getAgastScore(x_left, y_top + dy + 1, 1));
  for (int X = 1; X <= dx; X++)
  {
    ret_val += r_y1_i * int(l.getAgastScore(x_left + X, y_top + dy + 1, 1));
  }
  ret_val += C * int(l.getAgastScore(x_left + dx + 1, y_top + dy + 1, 1));

  return ((ret_val + scaling2 / 2) / scaling2);
}



//直接移植
/***
 * 2维平面的最大值抑制
 * @param layer
 * @param x_layer
 * @param y_layer
 * @return
 */
__device__ inline bool
BriskScaleSpace::isMax2D(BriskLayerOne* layers,const int layer, const int x_layer, const int y_layer)
{
  const PtrStepSzi& scores = layers[layer].scores();
  const int scorescols = scores.cols;
  const int* data = scores.ptr() + y_layer * scorescols + x_layer;
  // decision tree:
  const unsigned char center = (*data);
  data--;
  const unsigned char s_10 = *data;
  if (center < s_10)
    return false;
  data += 2;
  const unsigned char s10 = *data;
  if (center < s10)
    return false;
  data -= (scorescols + 1);
  const unsigned char s0_1 = *data;
  if (center < s0_1)
    return false;
  data += 2 * scorescols;
  const unsigned char s01 = *data;
  if (center < s01)
    return false;
  data--;
  const unsigned char s_11 = *data;
  if (center < s_11)
    return false;
  data += 2;
  const unsigned char s11 = *data;
  if (center < s11)
    return false;
  data -= 2 * scorescols;
  const unsigned char s1_1 = *data;
  if (center < s1_1)
    return false;
  data -= 2;
  const unsigned char s_1_1 = *data;
  if (center < s_1_1)
    return false;

  //对相等情况的特殊处理
  // reject neighbor maxima
  std::vector<int> delta;
  // put together a list of 2d-offsets to where the maximum is also reached
  if (center == s_1_1)
  {
    delta.push_back(-1);
    delta.push_back(-1);
  }
  if (center == s0_1)
  {
    delta.push_back(0);
    delta.push_back(-1);
  }
  if (center == s1_1)
  {
    delta.push_back(1);
    delta.push_back(-1);
  }
  if (center == s_10)
  {
    delta.push_back(-1);
    delta.push_back(0);
  }
  if (center == s10)
  {
    delta.push_back(1);
    delta.push_back(0);
  }
  if (center == s_11)
  {
    delta.push_back(-1);
    delta.push_back(1);
  }
  if (center == s01)
  {
    delta.push_back(0);
    delta.push_back(1);
  }
  if (center == s11)
  {
    delta.push_back(1);
    delta.push_back(1);
  }
  const unsigned int deltasize = (unsigned int)delta.size();
  if (deltasize != 0)
  {
    // in this case, we have to analyze the situation more carefully:
    // the values are gaussian blurred and then we really decide
    data = scores.ptr() + y_layer * scorescols + x_layer;
    int smoothedcenter = 4 * center + 2 * (s_10 + s10 + s0_1 + s01) + s_1_1 + s1_1 + s_11 + s11;
    for (unsigned int i = 0; i < deltasize; i += 2)
    {
      data = scores.ptr() + (y_layer - 1 + delta[i + 1]) * scorescols + x_layer + delta[i] - 1;
      int othercenter = *data;
      data++;
      othercenter += 2 * (*data);
      data++;
      othercenter += *data;
      data += scorescols;
      othercenter += 2 * (*data);
      data--;
      othercenter += 4 * (*data);
      data--;
      othercenter += 2 * (*data);
      data += scorescols;
      othercenter += *data;
      data++;
      othercenter += 2 * (*data);
      data++;
      othercenter += *data;
      if (othercenter > smoothedcenter)
        return false;
    }
  }
  return true;
}





/***
 * 直接进行3维空间里的最大值抑制
 * 整合了2Dsub-pixl和1d refine
 * @param layer
 * @param x_layer
 * @param y_layer
 * @param x 返回值，x的拟合值
 * @param y 返回值，y的拟合值
 * @param scale 返回值，scale的max值
 * @param ismax 返回值，是否是局部最大
 * @return
 */
// 3D maximum refinement centered around (x_layer,y_layer)
__device__ inline float
BriskScaleSpace::refine3D(BriskLayerOne* layers,const int layer, const int x_layer, const int y_layer, float& x, float& y, float& scale,
                          bool& ismax) const
{
  ismax = true;
  const BriskLayerOne& thisLayer = layers[layer];
  const int center = thisLayer.getAgastScore(x_layer, y_layer, 1);

  // check and get above maximum:
  float delta_x_above = 0, delta_y_above = 0;
  float max_above = getScoreMaxAbove(layers, layer, x_layer, y_layer, center, ismax, delta_x_above, delta_y_above);

  if (!ismax)
    return 0.0f;

  float max; // to be returned

  if (layer % 2 == 0)
  { // on octave
    // treat the patch below:
    float delta_x_below, delta_y_below;
    float max_below_float;
    int max_below = 0;
    if (layer == 0)
    {
      // guess the lower intra octave...
      const BriskLayerOne& l = layers[0];
      int s_0_0 = l.getAgastScore_5_8(x_layer - 1, y_layer - 1, 1);
      max_below = s_0_0;
      int s_1_0 = l.getAgastScore_5_8(x_layer, y_layer - 1, 1);
      max_below = std::max(s_1_0, max_below);
      int s_2_0 = l.getAgastScore_5_8(x_layer + 1, y_layer - 1, 1);
      max_below = std::max(s_2_0, max_below);
      int s_2_1 = l.getAgastScore_5_8(x_layer + 1, y_layer, 1);
      max_below = std::max(s_2_1, max_below);
      int s_1_1 = l.getAgastScore_5_8(x_layer, y_layer, 1);
      max_below = std::max(s_1_1, max_below);
      int s_0_1 = l.getAgastScore_5_8(x_layer - 1, y_layer, 1);
      max_below = std::max(s_0_1, max_below);
      int s_0_2 = l.getAgastScore_5_8(x_layer - 1, y_layer + 1, 1);
      max_below = std::max(s_0_2, max_below);
      int s_1_2 = l.getAgastScore_5_8(x_layer, y_layer + 1, 1);
      max_below = std::max(s_1_2, max_below);
      int s_2_2 = l.getAgastScore_5_8(x_layer + 1, y_layer + 1, 1);
      max_below = std::max(s_2_2, max_below);

      max_below_float = subpixel2D(s_0_0, s_0_1, s_0_2, s_1_0, s_1_1, s_1_2, s_2_0, s_2_1, s_2_2, delta_x_below,
                                   delta_y_below);
      max_below_float = (float)max_below;
    }
    else
    {
      max_below_float = getScoreMaxBelow(layers, layer, x_layer, y_layer, center, ismax, delta_x_below, delta_y_below);
      if (!ismax)
        return 0;
    }

    // get the patch on this layer:
    int s_0_0 = thisLayer.getAgastScore(x_layer - 1, y_layer - 1, 1);
    int s_1_0 = thisLayer.getAgastScore(x_layer, y_layer - 1, 1);
    int s_2_0 = thisLayer.getAgastScore(x_layer + 1, y_layer - 1, 1);
    int s_2_1 = thisLayer.getAgastScore(x_layer + 1, y_layer, 1);
    int s_1_1 = thisLayer.getAgastScore(x_layer, y_layer, 1);
    int s_0_1 = thisLayer.getAgastScore(x_layer - 1, y_layer, 1);
    int s_0_2 = thisLayer.getAgastScore(x_layer - 1, y_layer + 1, 1);
    int s_1_2 = thisLayer.getAgastScore(x_layer, y_layer + 1, 1);
    int s_2_2 = thisLayer.getAgastScore(x_layer + 1, y_layer + 1, 1);
    float delta_x_layer, delta_y_layer;
    float max_layer = subpixel2D(s_0_0, s_0_1, s_0_2, s_1_0, s_1_1, s_1_2, s_2_0, s_2_1, s_2_2, delta_x_layer,
                                 delta_y_layer);

    // calculate the relative scale (1D maximum):
    if (layer == 0)
    {
      scale = refine1D_2(max_below_float, std::max(float(center), max_layer), max_above, max);
    }
    else
      scale = refine1D(max_below_float, std::max(float(center), max_layer), max_above, max);

    if (scale > 1.0)
    {
      // interpolate the position:
      const float r0 = (1.5f - scale) / .5f;
      const float r1 = 1.0f - r0;
      x = (r0 * delta_x_layer + r1 * delta_x_above + float(x_layer)) * thisLayer.scale() + thisLayer.offset();
      y = (r0 * delta_y_layer + r1 * delta_y_above + float(y_layer)) * thisLayer.scale() + thisLayer.offset();
    }
    else
    {
      if (layer == 0)
      {
        // interpolate the position:
        const float r0 = (scale - 0.5f) / 0.5f;
        const float r_1 = 1.0f - r0;
        x = r0 * delta_x_layer + r_1 * delta_x_below + float(x_layer);
        y = r0 * delta_y_layer + r_1 * delta_y_below + float(y_layer);
      }
      else
      {
        // interpolate the position:
        const float r0 = (scale - 0.75f) / 0.25f;
        const float r_1 = 1.0f - r0;
        x = (r0 * delta_x_layer + r_1 * delta_x_below + float(x_layer)) * thisLayer.scale() + thisLayer.offset();
        y = (r0 * delta_y_layer + r_1 * delta_y_below + float(y_layer)) * thisLayer.scale() + thisLayer.offset();
      }
    }
  }
  else
  {
    // on intra
    // check the patch below:
    float delta_x_below, delta_y_below;
    float max_below = getScoreMaxBelow(layers,layer, x_layer, y_layer, center, ismax, delta_x_below, delta_y_below);
    if (!ismax)
      return 0.0f;

    // get the patch on this layer:
    int s_0_0 = thisLayer.getAgastScore(x_layer - 1, y_layer - 1, 1);
    int s_1_0 = thisLayer.getAgastScore(x_layer, y_layer - 1, 1);
    int s_2_0 = thisLayer.getAgastScore(x_layer + 1, y_layer - 1, 1);
    int s_2_1 = thisLayer.getAgastScore(x_layer + 1, y_layer, 1);
    int s_1_1 = thisLayer.getAgastScore(x_layer, y_layer, 1);
    int s_0_1 = thisLayer.getAgastScore(x_layer - 1, y_layer, 1);
    int s_0_2 = thisLayer.getAgastScore(x_layer - 1, y_layer + 1, 1);
    int s_1_2 = thisLayer.getAgastScore(x_layer, y_layer + 1, 1);
    int s_2_2 = thisLayer.getAgastScore(x_layer + 1, y_layer + 1, 1);
    float delta_x_layer, delta_y_layer;
    float max_layer = subpixel2D(s_0_0, s_0_1, s_0_2, s_1_0, s_1_1, s_1_2, s_2_0, s_2_1, s_2_2, delta_x_layer,
                                 delta_y_layer);

    // calculate the relative scale (1D maximum):
    scale = refine1D_1(max_below, std::max(float(center), max_layer), max_above, max);
    if (scale > 1.0)
    {
      // interpolate the position:
      const float r0 = 4.0f - scale * 3.0f;
      const float r1 = 1.0f - r0;
      x = (r0 * delta_x_layer + r1 * delta_x_above + float(x_layer)) * thisLayer.scale() + thisLayer.offset();
      y = (r0 * delta_y_layer + r1 * delta_y_above + float(y_layer)) * thisLayer.scale() + thisLayer.offset();
    }
    else
    {
      // interpolate the position:
      const float r0 = scale * 3.0f - 2.0f;
      const float r_1 = 1.0f - r0;
      x = (r0 * delta_x_layer + r_1 * delta_x_below + float(x_layer)) * thisLayer.scale() + thisLayer.offset();
      y = (r0 * delta_y_layer + r_1 * delta_y_below + float(y_layer)) * thisLayer.scale() + thisLayer.offset();
    }
  }

  // calculate the absolute scale:
  scale *= thisLayer.scale();

  // that's it, return the refined maximum:
  return max;
}


/***
 *直接移植？
 */
// return the maximum of score patches above or below
__device__ inline float
BriskScaleSpace::getScoreMaxAbove(BriskLayerOne* layers,const int layer, const int x_layer, const int y_layer, const int threshold,
                                  bool& ismax, float& dx, float& dy) const
{

  ismax = false;
  // relevant floating point coordinates
  float x_1;
  float x1;
  float y_1;
  float y1;

  // the layer above
  assert(layer + 1 < layers_);
  const BriskLayerOne& layerAbove = layers[layer + 1];

  if (layer % 2 == 0)
  {
    // octave
    x_1 = float(4 * (x_layer) - 1 - 2) / 6.0f;
    x1 = float(4 * (x_layer) - 1 + 2) / 6.0f;
    y_1 = float(4 * (y_layer) - 1 - 2) / 6.0f;
    y1 = float(4 * (y_layer) - 1 + 2) / 6.0f;
  }
  else
  {
    // intra
    x_1 = float(6 * (x_layer) - 1 - 3) / 8.0f;
    x1 = float(6 * (x_layer) - 1 + 3) / 8.0f;
    y_1 = float(6 * (y_layer) - 1 - 3) / 8.0f;
    y1 = float(6 * (y_layer) - 1 + 3) / 8.0f;
  }

  // check the first row
  int max_x = (int)x_1 + 1;
  int max_y = (int)y_1 + 1;
  float tmp_max;
  float maxval = (float)layerAbove.getAgastScore(x_1, y_1, 1);
  if (maxval > threshold)
    return 0;
  for (int x = (int)x_1 + 1; x <= int(x1); x++)
  {
    tmp_max = (float)layerAbove.getAgastScore(float(x), y_1, 1);
    if (tmp_max > threshold)
      return 0;
    if (tmp_max > maxval)
    {
      maxval = tmp_max;
      max_x = x;
    }
  }
  tmp_max = (float)layerAbove.getAgastScore(x1, y_1, 1);
  if (tmp_max > threshold)
    return 0;
  if (tmp_max > maxval)
  {
    maxval = tmp_max;
    max_x = int(x1);
  }

  // middle rows
  for (int y = (int)y_1 + 1; y <= int(y1); y++)
  {
    tmp_max = (float)layerAbove.getAgastScore(x_1, float(y), 1);
    if (tmp_max > threshold)
      return 0;
    if (tmp_max > maxval)
    {
      maxval = tmp_max;
      max_x = int(x_1 + 1);
      max_y = y;
    }
    for (int x = (int)x_1 + 1; x <= int(x1); x++)
    {
      tmp_max = (float)layerAbove.getAgastScore(x, y, 1);
      if (tmp_max > threshold)
        return 0;
      if (tmp_max > maxval)
      {
        maxval = tmp_max;
        max_x = x;
        max_y = y;
      }
    }
    tmp_max = (float)layerAbove.getAgastScore(x1, float(y), 1);
    if (tmp_max > threshold)
      return 0;
    if (tmp_max > maxval)
    {
      maxval = tmp_max;
      max_x = int(x1);
      max_y = y;
    }
  }

  // bottom row
  tmp_max = (float)layerAbove.getAgastScore(x_1, y1, 1);
  if (tmp_max > maxval)
  {
    maxval = tmp_max;
    max_x = int(x_1 + 1);
    max_y = int(y1);
  }
  for (int x = (int)x_1 + 1; x <= int(x1); x++)
  {
    tmp_max = (float)layerAbove.getAgastScore(float(x), y1, 1);
    if (tmp_max > maxval)
    {
      maxval = tmp_max;
      max_x = x;
      max_y = int(y1);
    }
  }
  tmp_max = (float)layerAbove.getAgastScore(x1, y1, 1);
  if (tmp_max > maxval)
  {
    maxval = tmp_max;
    max_x = int(x1);
    max_y = int(y1);
  }

  //find dx/dy:
  int s_0_0 = layerAbove.getAgastScore(max_x - 1, max_y - 1, 1);
  int s_1_0 = layerAbove.getAgastScore(max_x, max_y - 1, 1);
  int s_2_0 = layerAbove.getAgastScore(max_x + 1, max_y - 1, 1);
  int s_2_1 = layerAbove.getAgastScore(max_x + 1, max_y, 1);
  int s_1_1 = layerAbove.getAgastScore(max_x, max_y, 1);
  int s_0_1 = layerAbove.getAgastScore(max_x - 1, max_y, 1);
  int s_0_2 = layerAbove.getAgastScore(max_x - 1, max_y + 1, 1);
  int s_1_2 = layerAbove.getAgastScore(max_x, max_y + 1, 1);
  int s_2_2 = layerAbove.getAgastScore(max_x + 1, max_y + 1, 1);
  float dx_1, dy_1;
  float refined_max = subpixel2D(s_0_0, s_0_1, s_0_2, s_1_0, s_1_1, s_1_2, s_2_0, s_2_1, s_2_2, dx_1, dy_1);

  // calculate dx/dy in above coordinates
  float real_x = float(max_x) + dx_1;
  float real_y = float(max_y) + dy_1;
  bool returnrefined = true;
  if (layer % 2 == 0)
  {
    dx = (real_x * 6.0f + 1.0f) / 4.0f - float(x_layer);
    dy = (real_y * 6.0f + 1.0f) / 4.0f - float(y_layer);
  }
  else
  {
    dx = (real_x * 8.0f + 1.0f) / 6.0f - float(x_layer);
    dy = (real_y * 8.0f + 1.0f) / 6.0f - float(y_layer);
  }

  // saturate
  if (dx > 1.0f)
  {
    dx = 1.0f;
    returnrefined = false;
  }
  if (dx < -1.0f)
  {
    dx = -1.0f;
    returnrefined = false;
  }
  if (dy > 1.0f)
  {
    dy = 1.0f;
    returnrefined = false;
  }
  if (dy < -1.0f)
  {
    dy = -1.0f;
    returnrefined = false;
  }

  // done and ok.
  ismax = true;
  if (returnrefined)
  {
    return std::max(refined_max, maxval);
  }
  return maxval;
}


__device__ inline float
BriskScaleSpace::getScoreMaxBelow(BriskLayerOne* layers, const int layer, const int x_layer, const int y_layer, const int threshold,
                                  bool& ismax, float& dx, float& dy) const
{
  ismax = false;

  // relevant floating point coordinates
  float x_1;
  float x1;
  float y_1;
  float y1;

  if (layer % 2 == 0)
  {
    // octave
    x_1 = float(8 * (x_layer) + 1 - 4) / 6.0f;
    x1 = float(8 * (x_layer) + 1 + 4) / 6.0f;
    y_1 = float(8 * (y_layer) + 1 - 4) / 6.0f;
    y1 = float(8 * (y_layer) + 1 + 4) / 6.0f;
  }
  else
  {
    x_1 = float(6 * (x_layer) + 1 - 3) / 4.0f;
    x1 = float(6 * (x_layer) + 1 + 3) / 4.0f;
    y_1 = float(6 * (y_layer) + 1 - 3) / 4.0f;
    y1 = float(6 * (y_layer) + 1 + 3) / 4.0f;
  }

  // the layer below
  assert(layer > 0);
  const BriskLayerOne& layerBelow = layers[layer - 1];

  // check the first row
  int max_x = (int)x_1 + 1;
  int max_y = (int)y_1 + 1;
  float tmp_max;
  float max = (float)layerBelow.getAgastScore(x_1, y_1, 1);
  if (max > threshold)
    return 0;
  for (int x = (int)x_1 + 1; x <= int(x1); x++)
  {
    tmp_max = (float)layerBelow.getAgastScore(float(x), y_1, 1);
    if (tmp_max > threshold)
      return 0;
    if (tmp_max > max)
    {
      max = tmp_max;
      max_x = x;
    }
  }
  tmp_max = (float)layerBelow.getAgastScore(x1, y_1, 1);
  if (tmp_max > threshold)
    return 0;
  if (tmp_max > max)
  {
    max = tmp_max;
    max_x = int(x1);
  }

  // middle rows
  for (int y = (int)y_1 + 1; y <= int(y1); y++)
  {
    tmp_max = (float)layerBelow.getAgastScore(x_1, float(y), 1);
    if (tmp_max > threshold)
      return 0;
    if (tmp_max > max)
    {
      max = tmp_max;
      max_x = int(x_1 + 1);
      max_y = y;
    }
    for (int x = (int)x_1 + 1; x <= int(x1); x++)
    {
      tmp_max = (float)layerBelow.getAgastScore(x, y, 1);
      if (tmp_max > threshold)
        return 0;
      if (tmp_max == max)
      {
        const int t1 = 2
            * (layerBelow.getAgastScore(x - 1, y, 1) + layerBelow.getAgastScore(x + 1, y, 1)
               + layerBelow.getAgastScore(x, y + 1, 1) + layerBelow.getAgastScore(x, y - 1, 1))
                       + (layerBelow.getAgastScore(x + 1, y + 1, 1) + layerBelow.getAgastScore(x - 1, y + 1, 1)
                          + layerBelow.getAgastScore(x + 1, y - 1, 1) + layerBelow.getAgastScore(x - 1, y - 1, 1));
        const int t2 = 2
            * (layerBelow.getAgastScore(max_x - 1, max_y, 1) + layerBelow.getAgastScore(max_x + 1, max_y, 1)
               + layerBelow.getAgastScore(max_x, max_y + 1, 1) + layerBelow.getAgastScore(max_x, max_y - 1, 1))
                       + (layerBelow.getAgastScore(max_x + 1, max_y + 1, 1) + layerBelow.getAgastScore(max_x - 1,
                                                                                                       max_y + 1, 1)
                          + layerBelow.getAgastScore(max_x + 1, max_y - 1, 1)
                          + layerBelow.getAgastScore(max_x - 1, max_y - 1, 1));
        if (t1 > t2)
        {
          max_x = x;
          max_y = y;
        }
      }
      if (tmp_max > max)
      {
        max = tmp_max;
        max_x = x;
        max_y = y;
      }
    }
    tmp_max = (float)layerBelow.getAgastScore(x1, float(y), 1);
    if (tmp_max > threshold)
      return 0;
    if (tmp_max > max)
    {
      max = tmp_max;
      max_x = int(x1);
      max_y = y;
    }
  }

  // bottom row
  tmp_max = (float)layerBelow.getAgastScore(x_1, y1, 1);
  if (tmp_max > max)
  {
    max = tmp_max;
    max_x = int(x_1 + 1);
    max_y = int(y1);
  }
  for (int x = (int)x_1 + 1; x <= int(x1); x++)
  {
    tmp_max = (float)layerBelow.getAgastScore(float(x), y1, 1);
    if (tmp_max > max)
    {
      max = tmp_max;
      max_x = x;
      max_y = int(y1);
    }
  }
  tmp_max = (float)layerBelow.getAgastScore(x1, y1, 1);
  if (tmp_max > max)
  {
    max = tmp_max;
    max_x = int(x1);
    max_y = int(y1);
  }

  //find dx/dy:
  int s_0_0 = layerBelow.getAgastScore(max_x - 1, max_y - 1, 1);
  int s_1_0 = layerBelow.getAgastScore(max_x, max_y - 1, 1);
  int s_2_0 = layerBelow.getAgastScore(max_x + 1, max_y - 1, 1);
  int s_2_1 = layerBelow.getAgastScore(max_x + 1, max_y, 1);
  int s_1_1 = layerBelow.getAgastScore(max_x, max_y, 1);
  int s_0_1 = layerBelow.getAgastScore(max_x - 1, max_y, 1);
  int s_0_2 = layerBelow.getAgastScore(max_x - 1, max_y + 1, 1);
  int s_1_2 = layerBelow.getAgastScore(max_x, max_y + 1, 1);
  int s_2_2 = layerBelow.getAgastScore(max_x + 1, max_y + 1, 1);
  float dx_1, dy_1;
  float refined_max = subpixel2D(s_0_0, s_0_1, s_0_2, s_1_0, s_1_1, s_1_2, s_2_0, s_2_1, s_2_2, dx_1, dy_1);

  // calculate dx/dy in above coordinates
  float real_x = float(max_x) + dx_1;
  float real_y = float(max_y) + dy_1;
  bool returnrefined = true;
  if (layer % 2 == 0)
  {
    dx = (float)((real_x * 6.0 + 1.0) / 8.0) - float(x_layer);
    dy = (float)((real_y * 6.0 + 1.0) / 8.0) - float(y_layer);
  }
  else
  {
    dx = (float)((real_x * 4.0 - 1.0) / 6.0) - float(x_layer);
    dy = (float)((real_y * 4.0 - 1.0) / 6.0) - float(y_layer);
  }

  // saturate
  if (dx > 1.0)
  {
    dx = 1.0f;
    returnrefined = false;
  }
  if (dx < -1.0f)
  {
    dx = -1.0f;
    returnrefined = false;
  }
  if (dy > 1.0f)
  {
    dy = 1.0f;
    returnrefined = false;
  }
  if (dy < -1.0f)
  {
    dy = -1.0f;
    returnrefined = false;
  }

  // done and ok.
  ismax = true;
  if (returnrefined)
  {
    return std::max(refined_max, max);
  }
  return max;
}


//直接移植
/***
 * 定系数2次函数差值样本1,
 * 此时的二次函数y1，y2,y3值已经给定，x1,x2,x3由调用时上下层的前后位置关系决定
 * @param s_05
 * @param s0
 * @param s05
 * @param max
 * @return
 */
__device__ inline float
BriskScaleSpace::refine1D(const float s_05, const float s0, const float s05, float& max) const
{
  int i_05 = int(1024.0 * s_05 + 0.5);
  int i0 = int(1024.0 * s0 + 0.5);
  int i05 = int(1024.0 * s05 + 0.5);

  //   16.0000  -24.0000    8.0000//反推a公式
  //  -40.0000   54.0000  -14.0000//反推b公式
  //   24.0000  -27.0000    6.0000//反推c公式

  int three_a = 16 * i_05 - 24 * i0 + 8 * i05;
  // second derivative must be negative:
  if (three_a >= 0)
  {
    if (s0 >= s_05 && s0 >= s05)
    {
      max = s0;
      return 1.0f;
    }
    if (s_05 >= s0 && s_05 >= s05)
    {
      max = s_05;
      return 0.75f;
    }
    if (s05 >= s0 && s05 >= s_05)
    {
      max = s05;
      return 1.5f;
    }
  }

  int three_b = -40 * i_05 + 54 * i0 - 14 * i05;
  // calculate max location:
  float ret_val = -float(three_b) / float(2 * three_a);
  // saturate and return
  if (ret_val < 0.75)
    ret_val = 0.75;
  else if (ret_val > 1.5)
    ret_val = 1.5; // allow to be slightly off bounds ...?
  int three_c = +24 * i_05 - 27 * i0 + 6 * i05;
  max = float(three_c) + float(three_a) * ret_val * ret_val + float(three_b) * ret_val;
  max /= 3072.0f;
  return ret_val;
}


//直接移植
/***
 * 定系数2次函数差值样本1
 * @param s_05
 * @param s0
 * @param s05
 * @param max
 * @return
 */
__device__ inline float
BriskScaleSpace::refine1D_1(const float s_05, const float s0, const float s05, float& max) const
{
  int i_05 = int(1024.0 * s_05 + 0.5);
  int i0 = int(1024.0 * s0 + 0.5);
  int i05 = int(1024.0 * s05 + 0.5);

  //  4.5000   -9.0000    4.5000
  //-10.5000   18.0000   -7.5000
  //  6.0000   -8.0000    3.0000

  int two_a = 9 * i_05 - 18 * i0 + 9 * i05;
  // second derivative must be negative:
  if (two_a >= 0)
  {
    if (s0 >= s_05 && s0 >= s05)
    {
      max = s0;
      return 1.0f;
    }
    if (s_05 >= s0 && s_05 >= s05)
    {
      max = s_05;
      return 0.6666666666666666666666666667f;
    }
    if (s05 >= s0 && s05 >= s_05)
    {
      max = s05;
      return 1.3333333333333333333333333333f;
    }
  }

  int two_b = -21 * i_05 + 36 * i0 - 15 * i05;
  // calculate max location:
  float ret_val = -float(two_b) / float(2 * two_a);
  // saturate and return
  if (ret_val < 0.6666666666666666666666666667f)
    ret_val = 0.666666666666666666666666667f;
  else if (ret_val > 1.33333333333333333333333333f)
    ret_val = 1.333333333333333333333333333f;
  int two_c = +12 * i_05 - 16 * i0 + 6 * i05;
  max = float(two_c) + float(two_a) * ret_val * ret_val + float(two_b) * ret_val;
  max /= 2048.0f;
  return ret_val;
}

//直接移植
__device__ inline float
BriskScaleSpace::refine1D_2(const float s_05, const float s0, const float s05, float& max) const
{
  int i_05 = int(1024.0 * s_05 + 0.5);
  int i0 = int(1024.0 * s0 + 0.5);
  int i05 = int(1024.0 * s05 + 0.5);

  //   18.0000  -30.0000   12.0000
  //  -45.0000   65.0000  -20.0000
  //   27.0000  -30.0000    8.0000

  int a = 2 * i_05 - 4 * i0 + 2 * i05;
  // second derivative must be negative:
  if (a >= 0)
  {
    if (s0 >= s_05 && s0 >= s05)
    {
      max = s0;
      return 1.0f;
    }
    if (s_05 >= s0 && s_05 >= s05)
    {
      max = s_05;
      return 0.7f;
    }
    if (s05 >= s0 && s05 >= s_05)
    {
      max = s05;
      return 1.5f;
    }
  }

  int b = -5 * i_05 + 8 * i0 - 3 * i05;
  // calculate max location:
  float ret_val = -float(b) / float(2 * a);
  // saturate and return
  if (ret_val < 0.7f)
    ret_val = 0.7f;
  else if (ret_val > 1.5f)
    ret_val = 1.5f; // allow to be slightly off bounds ...?
  int c = +3 * i_05 - 3 * i0 + 1 * i05;
  max = float(c) + float(a) * ret_val * ret_val + float(b) * ret_val;
  max /= 1024;
  return ret_val;
}


//直接移植
/***
 * 猜想：9个像素的方格不知道干啥
 * 反正返回的是一个9点插值亮度？
 * 何必呢
 * @param s_0_0
 * @param s_0_1
 * @param s_0_2
 * @param s_1_0
 * @param s_1_1
 * @param s_1_2
 * @param s_2_0
 * @param s_2_1
 * @param s_2_2
 * @param delta_x
 * @param delta_y
 * @return
 */
__device__ inline float
BriskScaleSpace::subpixel2D(const int s_0_0, const int s_0_1, const int s_0_2, const int s_1_0, const int s_1_1,
                            const int s_1_2, const int s_2_0, const int s_2_1, const int s_2_2, float& delta_x,
                            float& delta_y) const
{

  // the coefficients of the 2d quadratic function least-squares fit:
  int tmp1 = s_0_0 + s_0_2 - 2 * s_1_1 + s_2_0 + s_2_2;
  int coeff1 = 3 * (tmp1 + s_0_1 - ((s_1_0 + s_1_2) << 1) + s_2_1);
  int coeff2 = 3 * (tmp1 - ((s_0_1 + s_2_1) << 1) + s_1_0 + s_1_2);
  int tmp2 = s_0_2 - s_2_0;
  int tmp3 = (s_0_0 + tmp2 - s_2_2);
  int tmp4 = tmp3 - 2 * tmp2;
  int coeff3 = -3 * (tmp3 + s_0_1 - s_2_1);
  int coeff4 = -3 * (tmp4 + s_1_0 - s_1_2);
  int coeff5 = (s_0_0 - s_0_2 - s_2_0 + s_2_2) << 2;
  int coeff6 = -(s_0_0 + s_0_2 - ((s_1_0 + s_0_1 + s_1_2 + s_2_1) << 1) - 5 * s_1_1 + s_2_0 + s_2_2) << 1;

  // 2nd derivative test:
  int H_det = 4 * coeff1 * coeff2 - coeff5 * coeff5;

  if (H_det == 0)
  {
    delta_x = 0.0f;
    delta_y = 0.0f;
    return float(coeff6) / 18.0f;
  }

  if (!(H_det > 0 && coeff1 < 0))
  {
    // The maximum must be at the one of the 4 patch corners.
    int tmp_max = coeff3 + coeff4 + coeff5;
    delta_x = 1.0f;
    delta_y = 1.0f;

    int tmp = -coeff3 + coeff4 - coeff5;
    if (tmp > tmp_max)
    {
      tmp_max = tmp;
      delta_x = -1.0f;
      delta_y = 1.0f;
    }
    tmp = coeff3 - coeff4 - coeff5;
    if (tmp > tmp_max)
    {
      tmp_max = tmp;
      delta_x = 1.0f;
      delta_y = -1.0f;
    }
    tmp = -coeff3 - coeff4 + coeff5;
    if (tmp > tmp_max)
    {
      tmp_max = tmp;
      delta_x = -1.0f;
      delta_y = -1.0f;
    }
    return float(tmp_max + coeff1 + coeff2 + coeff6) / 18.0f;
  }

  // this is hopefully the normal outcome of the Hessian test
  delta_x = float(2 * coeff2 * coeff3 - coeff4 * coeff5) / float(-H_det);
  delta_y = float(2 * coeff1 * coeff4 - coeff3 * coeff5) / float(-H_det);
  // TODO: this is not correct, but easy, so perform a real boundary maximum search:
  bool tx = false;
  bool tx_ = false;
  bool ty = false;
  bool ty_ = false;
  if (delta_x > 1.0)
    tx = true;
  else if (delta_x < -1.0)
    tx_ = true;
  if (delta_y > 1.0)
    ty = true;
  if (delta_y < -1.0)
    ty_ = true;

  if (tx || tx_ || ty || ty_)
  {
    // get two candidates:
    float delta_x1 = 0.0f, delta_x2 = 0.0f, delta_y1 = 0.0f, delta_y2 = 0.0f;
    if (tx)
    {
      delta_x1 = 1.0f;
      delta_y1 = -float(coeff4 + coeff5) / float(2 * coeff2);
      if (delta_y1 > 1.0f)
        delta_y1 = 1.0f;
      else if (delta_y1 < -1.0f)
        delta_y1 = -1.0f;
    }
    else if (tx_)
    {
      delta_x1 = -1.0f;
      delta_y1 = -float(coeff4 - coeff5) / float(2 * coeff2);
      if (delta_y1 > 1.0f)
        delta_y1 = 1.0f;
      else if (delta_y1 < -1.0)
        delta_y1 = -1.0f;
    }
    if (ty)
    {
      delta_y2 = 1.0f;
      delta_x2 = -float(coeff3 + coeff5) / float(2 * coeff1);
      if (delta_x2 > 1.0f)
        delta_x2 = 1.0f;
      else if (delta_x2 < -1.0f)
        delta_x2 = -1.0f;
    }
    else if (ty_)
    {
      delta_y2 = -1.0f;
      delta_x2 = -float(coeff3 - coeff5) / float(2 * coeff1);
      if (delta_x2 > 1.0f)
        delta_x2 = 1.0f;
      else if (delta_x2 < -1.0f)
        delta_x2 = -1.0f;
    }
    // insert both options for evaluation which to pick
    float max1 = (coeff1 * delta_x1 * delta_x1 + coeff2 * delta_y1 * delta_y1 + coeff3 * delta_x1 + coeff4 * delta_y1
                  + coeff5 * delta_x1 * delta_y1 + coeff6)
                 / 18.0f;
    float max2 = (coeff1 * delta_x2 * delta_x2 + coeff2 * delta_y2 * delta_y2 + coeff3 * delta_x2 + coeff4 * delta_y2
                  + coeff5 * delta_x2 * delta_y2 + coeff6)
                 / 18.0f;
    if (max1 > max2)
    {
      delta_x = delta_x1;
      delta_y = delta_y1;
      return max1;
    }
    else
    {
      delta_x = delta_x2;
      delta_y = delta_y2;
      return max2;
    }
  }

  // this is the case of the maximum inside the boundaries:
  return (coeff1 * delta_x * delta_x + coeff2 * delta_y * delta_y + coeff3 * delta_x + coeff4 * delta_y
          + coeff5 * delta_x * delta_y + coeff6)
         / 18.0f;
}


