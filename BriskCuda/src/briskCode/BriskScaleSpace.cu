#include "hip/hip_runtime.h"

/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */


#include "BriskScaleSpace.cuh"



/***
 *直接移植？
 */
/***
 * 重点
 * @param layer
 * @param x_layer
 * @param y_layer
 * @param threshold
 * @param ismax
 * @param dx
 * @param dy
 * @return
 */



/***
 * 直接进行3维空间里的最大值抑制
 * 整合了2Dsub-pixl和1d refine
 * @param layer
 * @param x_layer
 * @param y_layer
 * @param x 返回值，x的拟合值
 * @param y 返回值，y的拟合值
 * @param scale 返回值，scale的max值
 * @param ismax 返回值，是否是局部最大
 * @return
 */
// 3D maximum refinement centered around (x_layer,y_layer)
__device__ inline float
BriskScaleSpace::refine3D(BriskLayerOne* layers,const int layer, const int x_layer, const int y_layer, float& x, float& y, float& scale,
                          bool& ismax) const
{
  ismax = true;
  const BriskLayerOne& thisLayer = layers[layer];
  const int center = thisLayer.getAgastScore(x_layer, y_layer, 1);

  // check and get above maximum:
  float delta_x_above = 0, delta_y_above = 0;
  float max_above = getScoreMaxAbove(layers, layer, x_layer, y_layer, center, ismax, delta_x_above, delta_y_above);

  if (!ismax)
    return 0.0f;

  float max; // to be returned

  if (layer % 2 == 0)
  { // on octave
    // treat the patch below:
    float delta_x_below, delta_y_below;
    float max_below_float;
    int max_below = 0;
    if (layer == 0)
    {
      // guess the lower intra octave...
      const BriskLayerOne& l = layers[0];
      int s_0_0 = l.getAgastScore_5_8(x_layer - 1, y_layer - 1, 1);
      max_below = s_0_0;
      int s_1_0 = l.getAgastScore_5_8(x_layer, y_layer - 1, 1);
      max_below = std::max(s_1_0, max_below);
      int s_2_0 = l.getAgastScore_5_8(x_layer + 1, y_layer - 1, 1);
      max_below = std::max(s_2_0, max_below);
      int s_2_1 = l.getAgastScore_5_8(x_layer + 1, y_layer, 1);
      max_below = std::max(s_2_1, max_below);
      int s_1_1 = l.getAgastScore_5_8(x_layer, y_layer, 1);
      max_below = std::max(s_1_1, max_below);
      int s_0_1 = l.getAgastScore_5_8(x_layer - 1, y_layer, 1);
      max_below = std::max(s_0_1, max_below);
      int s_0_2 = l.getAgastScore_5_8(x_layer - 1, y_layer + 1, 1);
      max_below = std::max(s_0_2, max_below);
      int s_1_2 = l.getAgastScore_5_8(x_layer, y_layer + 1, 1);
      max_below = std::max(s_1_2, max_below);
      int s_2_2 = l.getAgastScore_5_8(x_layer + 1, y_layer + 1, 1);
      max_below = std::max(s_2_2, max_below);

      max_below_float = subpixel2D(s_0_0, s_0_1, s_0_2, s_1_0, s_1_1, s_1_2, s_2_0, s_2_1, s_2_2, delta_x_below,
                                   delta_y_below);
      max_below_float = (float)max_below;
    }
    else
    {
      max_below_float = getScoreMaxBelow(layers, layer, x_layer, y_layer, center, ismax, delta_x_below, delta_y_below);
      if (!ismax)
        return 0;
    }

    // get the patch on this layer:
    int s_0_0 = thisLayer.getAgastScore(x_layer - 1, y_layer - 1, 1);
    int s_1_0 = thisLayer.getAgastScore(x_layer, y_layer - 1, 1);
    int s_2_0 = thisLayer.getAgastScore(x_layer + 1, y_layer - 1, 1);
    int s_2_1 = thisLayer.getAgastScore(x_layer + 1, y_layer, 1);
    int s_1_1 = thisLayer.getAgastScore(x_layer, y_layer, 1);
    int s_0_1 = thisLayer.getAgastScore(x_layer - 1, y_layer, 1);
    int s_0_2 = thisLayer.getAgastScore(x_layer - 1, y_layer + 1, 1);
    int s_1_2 = thisLayer.getAgastScore(x_layer, y_layer + 1, 1);
    int s_2_2 = thisLayer.getAgastScore(x_layer + 1, y_layer + 1, 1);
    float delta_x_layer, delta_y_layer;
    float max_layer = subpixel2D(s_0_0, s_0_1, s_0_2, s_1_0, s_1_1, s_1_2, s_2_0, s_2_1, s_2_2, delta_x_layer,
                                 delta_y_layer);

    // calculate the relative scale (1D maximum):
    if (layer == 0)
    {
      scale = refine1D_2(max_below_float, std::max(float(center), max_layer), max_above, max);
    }
    else
      scale = refine1D(max_below_float, std::max(float(center), max_layer), max_above, max);

    if (scale > 1.0)
    {
      // interpolate the position:
      const float r0 = (1.5f - scale) / .5f;
      const float r1 = 1.0f - r0;
      x = (r0 * delta_x_layer + r1 * delta_x_above + float(x_layer)) * thisLayer.scale() + thisLayer.offset();
      y = (r0 * delta_y_layer + r1 * delta_y_above + float(y_layer)) * thisLayer.scale() + thisLayer.offset();
    }
    else
    {
      if (layer == 0)
      {
        // interpolate the position:
        const float r0 = (scale - 0.5f) / 0.5f;
        const float r_1 = 1.0f - r0;
        x = r0 * delta_x_layer + r_1 * delta_x_below + float(x_layer);
        y = r0 * delta_y_layer + r_1 * delta_y_below + float(y_layer);
      }
      else
      {
        // interpolate the position:
        const float r0 = (scale - 0.75f) / 0.25f;
        const float r_1 = 1.0f - r0;
        x = (r0 * delta_x_layer + r_1 * delta_x_below + float(x_layer)) * thisLayer.scale() + thisLayer.offset();
        y = (r0 * delta_y_layer + r_1 * delta_y_below + float(y_layer)) * thisLayer.scale() + thisLayer.offset();
      }
    }
  }
  else
  {
    // on intra
    // check the patch below:
    float delta_x_below, delta_y_below;
    float max_below = getScoreMaxBelow(layers,layer, x_layer, y_layer, center, ismax, delta_x_below, delta_y_below);
    if (!ismax)
      return 0.0f;

    // get the patch on this layer:
    int s_0_0 = thisLayer.getAgastScore(x_layer - 1, y_layer - 1, 1);
    int s_1_0 = thisLayer.getAgastScore(x_layer, y_layer - 1, 1);
    int s_2_0 = thisLayer.getAgastScore(x_layer + 1, y_layer - 1, 1);
    int s_2_1 = thisLayer.getAgastScore(x_layer + 1, y_layer, 1);
    int s_1_1 = thisLayer.getAgastScore(x_layer, y_layer, 1);
    int s_0_1 = thisLayer.getAgastScore(x_layer - 1, y_layer, 1);
    int s_0_2 = thisLayer.getAgastScore(x_layer - 1, y_layer + 1, 1);
    int s_1_2 = thisLayer.getAgastScore(x_layer, y_layer + 1, 1);
    int s_2_2 = thisLayer.getAgastScore(x_layer + 1, y_layer + 1, 1);
    float delta_x_layer, delta_y_layer;
    float max_layer = subpixel2D(s_0_0, s_0_1, s_0_2, s_1_0, s_1_1, s_1_2, s_2_0, s_2_1, s_2_2, delta_x_layer,
                                 delta_y_layer);

    // calculate the relative scale (1D maximum):
    scale = refine1D_1(max_below, std::max(float(center), max_layer), max_above, max);
    if (scale > 1.0)
    {
      // interpolate the position:
      const float r0 = 4.0f - scale * 3.0f;
      const float r1 = 1.0f - r0;
      x = (r0 * delta_x_layer + r1 * delta_x_above + float(x_layer)) * thisLayer.scale() + thisLayer.offset();
      y = (r0 * delta_y_layer + r1 * delta_y_above + float(y_layer)) * thisLayer.scale() + thisLayer.offset();
    }
    else
    {
      // interpolate the position:
      const float r0 = scale * 3.0f - 2.0f;
      const float r_1 = 1.0f - r0;
      x = (r0 * delta_x_layer + r_1 * delta_x_below + float(x_layer)) * thisLayer.scale() + thisLayer.offset();
      y = (r0 * delta_y_layer + r_1 * delta_y_below + float(y_layer)) * thisLayer.scale() + thisLayer.offset();
    }
  }

  // calculate the absolute scale:
  scale *= thisLayer.scale();

  // that's it, return the refined maximum:
  return max;
}


/***
 *直接移植？
 */
// return the maximum of score patches above or below
__device__ inline float
BriskScaleSpace::getScoreMaxAbove(BriskLayerOne* layers,const int layer, const int x_layer, const int y_layer, const int threshold,
                                  bool& ismax, float& dx, float& dy) const
{

  ismax = false;
  // relevant floating point coordinates
  float x_1;
  float x1;
  float y_1;
  float y1;

  // the layer above
  assert(layer + 1 < layers_);
  const BriskLayerOne& layerAbove = layers[layer + 1];

  if (layer % 2 == 0)
  {
    // octave
    x_1 = float(4 * (x_layer) - 1 - 2) / 6.0f;
    x1 = float(4 * (x_layer) - 1 + 2) / 6.0f;
    y_1 = float(4 * (y_layer) - 1 - 2) / 6.0f;
    y1 = float(4 * (y_layer) - 1 + 2) / 6.0f;
  }
  else
  {
    // intra
    x_1 = float(6 * (x_layer) - 1 - 3) / 8.0f;
    x1 = float(6 * (x_layer) - 1 + 3) / 8.0f;
    y_1 = float(6 * (y_layer) - 1 - 3) / 8.0f;
    y1 = float(6 * (y_layer) - 1 + 3) / 8.0f;
  }

  // check the first row
  int max_x = (int)x_1 + 1;
  int max_y = (int)y_1 + 1;
  float tmp_max;
  float maxval = (float)layerAbove.getAgastScore(x_1, y_1, 1);
  if (maxval > threshold)
    return 0;
  for (int x = (int)x_1 + 1; x <= int(x1); x++)
  {
    tmp_max = (float)layerAbove.getAgastScore(float(x), y_1, 1);
    if (tmp_max > threshold)
      return 0;
    if (tmp_max > maxval)
    {
      maxval = tmp_max;
      max_x = x;
    }
  }
  tmp_max = (float)layerAbove.getAgastScore(x1, y_1, 1);
  if (tmp_max > threshold)
    return 0;
  if (tmp_max > maxval)
  {
    maxval = tmp_max;
    max_x = int(x1);
  }

  // middle rows
  for (int y = (int)y_1 + 1; y <= int(y1); y++)
  {
    tmp_max = (float)layerAbove.getAgastScore(x_1, float(y), 1);
    if (tmp_max > threshold)
      return 0;
    if (tmp_max > maxval)
    {
      maxval = tmp_max;
      max_x = int(x_1 + 1);
      max_y = y;
    }
    for (int x = (int)x_1 + 1; x <= int(x1); x++)
    {
      tmp_max = (float)layerAbove.getAgastScore(x, y, 1);
      if (tmp_max > threshold)
        return 0;
      if (tmp_max > maxval)
      {
        maxval = tmp_max;
        max_x = x;
        max_y = y;
      }
    }
    tmp_max = (float)layerAbove.getAgastScore(x1, float(y), 1);
    if (tmp_max > threshold)
      return 0;
    if (tmp_max > maxval)
    {
      maxval = tmp_max;
      max_x = int(x1);
      max_y = y;
    }
  }

  // bottom row
  tmp_max = (float)layerAbove.getAgastScore(x_1, y1, 1);
  if (tmp_max > maxval)
  {
    maxval = tmp_max;
    max_x = int(x_1 + 1);
    max_y = int(y1);
  }
  for (int x = (int)x_1 + 1; x <= int(x1); x++)
  {
    tmp_max = (float)layerAbove.getAgastScore(float(x), y1, 1);
    if (tmp_max > maxval)
    {
      maxval = tmp_max;
      max_x = x;
      max_y = int(y1);
    }
  }
  tmp_max = (float)layerAbove.getAgastScore(x1, y1, 1);
  if (tmp_max > maxval)
  {
    maxval = tmp_max;
    max_x = int(x1);
    max_y = int(y1);
  }

  //find dx/dy:
  int s_0_0 = layerAbove.getAgastScore(max_x - 1, max_y - 1, 1);
  int s_1_0 = layerAbove.getAgastScore(max_x, max_y - 1, 1);
  int s_2_0 = layerAbove.getAgastScore(max_x + 1, max_y - 1, 1);
  int s_2_1 = layerAbove.getAgastScore(max_x + 1, max_y, 1);
  int s_1_1 = layerAbove.getAgastScore(max_x, max_y, 1);
  int s_0_1 = layerAbove.getAgastScore(max_x - 1, max_y, 1);
  int s_0_2 = layerAbove.getAgastScore(max_x - 1, max_y + 1, 1);
  int s_1_2 = layerAbove.getAgastScore(max_x, max_y + 1, 1);
  int s_2_2 = layerAbove.getAgastScore(max_x + 1, max_y + 1, 1);
  float dx_1, dy_1;
  float refined_max = subpixel2D(s_0_0, s_0_1, s_0_2, s_1_0, s_1_1, s_1_2, s_2_0, s_2_1, s_2_2, dx_1, dy_1);

  // calculate dx/dy in above coordinates
  float real_x = float(max_x) + dx_1;
  float real_y = float(max_y) + dy_1;
  bool returnrefined = true;
  if (layer % 2 == 0)
  {
    dx = (real_x * 6.0f + 1.0f) / 4.0f - float(x_layer);
    dy = (real_y * 6.0f + 1.0f) / 4.0f - float(y_layer);
  }
  else
  {
    dx = (real_x * 8.0f + 1.0f) / 6.0f - float(x_layer);
    dy = (real_y * 8.0f + 1.0f) / 6.0f - float(y_layer);
  }

  // saturate
  if (dx > 1.0f)
  {
    dx = 1.0f;
    returnrefined = false;
  }
  if (dx < -1.0f)
  {
    dx = -1.0f;
    returnrefined = false;
  }
  if (dy > 1.0f)
  {
    dy = 1.0f;
    returnrefined = false;
  }
  if (dy < -1.0f)
  {
    dy = -1.0f;
    returnrefined = false;
  }

  // done and ok.
  ismax = true;
  if (returnrefined)
  {
    return std::max(refined_max, maxval);
  }
  return maxval;
}


__device__ inline float
BriskScaleSpace::getScoreMaxBelow(BriskLayerOne* layers, const int layer, const int x_layer, const int y_layer, const int threshold,
                                  bool& ismax, float& dx, float& dy) const
{
  ismax = false;

  // relevant floating point coordinates
  float x_1;
  float x1;
  float y_1;
  float y1;

  if (layer % 2 == 0)
  {
    // octave
    x_1 = float(8 * (x_layer) + 1 - 4) / 6.0f;
    x1 = float(8 * (x_layer) + 1 + 4) / 6.0f;
    y_1 = float(8 * (y_layer) + 1 - 4) / 6.0f;
    y1 = float(8 * (y_layer) + 1 + 4) / 6.0f;
  }
  else
  {
    x_1 = float(6 * (x_layer) + 1 - 3) / 4.0f;
    x1 = float(6 * (x_layer) + 1 + 3) / 4.0f;
    y_1 = float(6 * (y_layer) + 1 - 3) / 4.0f;
    y1 = float(6 * (y_layer) + 1 + 3) / 4.0f;
  }

  // the layer below
  assert(layer > 0);
  const BriskLayerOne& layerBelow = layers[layer - 1];

  // check the first row
  int max_x = (int)x_1 + 1;
  int max_y = (int)y_1 + 1;
  float tmp_max;
  float max = (float)layerBelow.getAgastScore(x_1, y_1, 1);
  if (max > threshold)
    return 0;
  for (int x = (int)x_1 + 1; x <= int(x1); x++)
  {
    tmp_max = (float)layerBelow.getAgastScore(float(x), y_1, 1);
    if (tmp_max > threshold)
      return 0;
    if (tmp_max > max)
    {
      max = tmp_max;
      max_x = x;
    }
  }
  tmp_max = (float)layerBelow.getAgastScore(x1, y_1, 1);
  if (tmp_max > threshold)
    return 0;
  if (tmp_max > max)
  {
    max = tmp_max;
    max_x = int(x1);
  }

  // middle rows
  for (int y = (int)y_1 + 1; y <= int(y1); y++)
  {
    tmp_max = (float)layerBelow.getAgastScore(x_1, float(y), 1);
    if (tmp_max > threshold)
      return 0;
    if (tmp_max > max)
    {
      max = tmp_max;
      max_x = int(x_1 + 1);
      max_y = y;
    }
    for (int x = (int)x_1 + 1; x <= int(x1); x++)
    {
      tmp_max = (float)layerBelow.getAgastScore(x, y, 1);
      if (tmp_max > threshold)
        return 0;
      if (tmp_max == max)
      {
        const int t1 = 2
            * (layerBelow.getAgastScore(x - 1, y, 1) + layerBelow.getAgastScore(x + 1, y, 1)
               + layerBelow.getAgastScore(x, y + 1, 1) + layerBelow.getAgastScore(x, y - 1, 1))
                       + (layerBelow.getAgastScore(x + 1, y + 1, 1) + layerBelow.getAgastScore(x - 1, y + 1, 1)
                          + layerBelow.getAgastScore(x + 1, y - 1, 1) + layerBelow.getAgastScore(x - 1, y - 1, 1));
        const int t2 = 2
            * (layerBelow.getAgastScore(max_x - 1, max_y, 1) + layerBelow.getAgastScore(max_x + 1, max_y, 1)
               + layerBelow.getAgastScore(max_x, max_y + 1, 1) + layerBelow.getAgastScore(max_x, max_y - 1, 1))
                       + (layerBelow.getAgastScore(max_x + 1, max_y + 1, 1) + layerBelow.getAgastScore(max_x - 1,
                                                                                                       max_y + 1, 1)
                          + layerBelow.getAgastScore(max_x + 1, max_y - 1, 1)
                          + layerBelow.getAgastScore(max_x - 1, max_y - 1, 1));
        if (t1 > t2)
        {
          max_x = x;
          max_y = y;
        }
      }
      if (tmp_max > max)
      {
        max = tmp_max;
        max_x = x;
        max_y = y;
      }
    }
    tmp_max = (float)layerBelow.getAgastScore(x1, float(y), 1);
    if (tmp_max > threshold)
      return 0;
    if (tmp_max > max)
    {
      max = tmp_max;
      max_x = int(x1);
      max_y = y;
    }
  }

  // bottom row
  tmp_max = (float)layerBelow.getAgastScore(x_1, y1, 1);
  if (tmp_max > max)
  {
    max = tmp_max;
    max_x = int(x_1 + 1);
    max_y = int(y1);
  }
  for (int x = (int)x_1 + 1; x <= int(x1); x++)
  {
    tmp_max = (float)layerBelow.getAgastScore(float(x), y1, 1);
    if (tmp_max > max)
    {
      max = tmp_max;
      max_x = x;
      max_y = int(y1);
    }
  }
  tmp_max = (float)layerBelow.getAgastScore(x1, y1, 1);
  if (tmp_max > max)
  {
    max = tmp_max;
    max_x = int(x1);
    max_y = int(y1);
  }

  //find dx/dy:
  int s_0_0 = layerBelow.getAgastScore(max_x - 1, max_y - 1, 1);
  int s_1_0 = layerBelow.getAgastScore(max_x, max_y - 1, 1);
  int s_2_0 = layerBelow.getAgastScore(max_x + 1, max_y - 1, 1);
  int s_2_1 = layerBelow.getAgastScore(max_x + 1, max_y, 1);
  int s_1_1 = layerBelow.getAgastScore(max_x, max_y, 1);
  int s_0_1 = layerBelow.getAgastScore(max_x - 1, max_y, 1);
  int s_0_2 = layerBelow.getAgastScore(max_x - 1, max_y + 1, 1);
  int s_1_2 = layerBelow.getAgastScore(max_x, max_y + 1, 1);
  int s_2_2 = layerBelow.getAgastScore(max_x + 1, max_y + 1, 1);
  float dx_1, dy_1;
  float refined_max = subpixel2D(s_0_0, s_0_1, s_0_2, s_1_0, s_1_1, s_1_2, s_2_0, s_2_1, s_2_2, dx_1, dy_1);

  // calculate dx/dy in above coordinates
  float real_x = float(max_x) + dx_1;
  float real_y = float(max_y) + dy_1;
  bool returnrefined = true;
  if (layer % 2 == 0)
  {
    dx = (float)((real_x * 6.0 + 1.0) / 8.0) - float(x_layer);
    dy = (float)((real_y * 6.0 + 1.0) / 8.0) - float(y_layer);
  }
  else
  {
    dx = (float)((real_x * 4.0 - 1.0) / 6.0) - float(x_layer);
    dy = (float)((real_y * 4.0 - 1.0) / 6.0) - float(y_layer);
  }

  // saturate
  if (dx > 1.0)
  {
    dx = 1.0f;
    returnrefined = false;
  }
  if (dx < -1.0f)
  {
    dx = -1.0f;
    returnrefined = false;
  }
  if (dy > 1.0f)
  {
    dy = 1.0f;
    returnrefined = false;
  }
  if (dy < -1.0f)
  {
    dy = -1.0f;
    returnrefined = false;
  }

  // done and ok.
  ismax = true;
  if (returnrefined)
  {
    return std::max(refined_max, max);
  }
  return max;
}


//直接移植
/***
 * 定系数2次函数差值样本1,
 * 此时的二次函数y1，y2,y3值已经给定，x1,x2,x3由调用时上下层的前后位置关系决定
 * @param s_05
 * @param s0
 * @param s05
 * @param max
 * @return
 */
__device__ inline float
BriskScaleSpace::refine1D(const float s_05, const float s0, const float s05, float& max) const
{
  int i_05 = int(1024.0 * s_05 + 0.5);
  int i0 = int(1024.0 * s0 + 0.5);
  int i05 = int(1024.0 * s05 + 0.5);

  //   16.0000  -24.0000    8.0000//反推a公式
  //  -40.0000   54.0000  -14.0000//反推b公式
  //   24.0000  -27.0000    6.0000//反推c公式

  int three_a = 16 * i_05 - 24 * i0 + 8 * i05;
  // second derivative must be negative:
  if (three_a >= 0)
  {
    if (s0 >= s_05 && s0 >= s05)
    {
      max = s0;
      return 1.0f;
    }
    if (s_05 >= s0 && s_05 >= s05)
    {
      max = s_05;
      return 0.75f;
    }
    if (s05 >= s0 && s05 >= s_05)
    {
      max = s05;
      return 1.5f;
    }
  }

  int three_b = -40 * i_05 + 54 * i0 - 14 * i05;
  // calculate max location:
  float ret_val = -float(three_b) / float(2 * three_a);
  // saturate and return
  if (ret_val < 0.75)
    ret_val = 0.75;
  else if (ret_val > 1.5)
    ret_val = 1.5; // allow to be slightly off bounds ...?
  int three_c = +24 * i_05 - 27 * i0 + 6 * i05;
  max = float(three_c) + float(three_a) * ret_val * ret_val + float(three_b) * ret_val;
  max /= 3072.0f;
  return ret_val;
}


//直接移植
/***
 * 定系数2次函数差值样本1
 * @param s_05
 * @param s0
 * @param s05
 * @param max
 * @return
 */
__device__ inline float
BriskScaleSpace::refine1D_1(const float s_05, const float s0, const float s05, float& max) const
{
  int i_05 = int(1024.0 * s_05 + 0.5);
  int i0 = int(1024.0 * s0 + 0.5);
  int i05 = int(1024.0 * s05 + 0.5);

  //  4.5000   -9.0000    4.5000
  //-10.5000   18.0000   -7.5000
  //  6.0000   -8.0000    3.0000

  int two_a = 9 * i_05 - 18 * i0 + 9 * i05;
  // second derivative must be negative:
  if (two_a >= 0)
  {
    if (s0 >= s_05 && s0 >= s05)
    {
      max = s0;
      return 1.0f;
    }
    if (s_05 >= s0 && s_05 >= s05)
    {
      max = s_05;
      return 0.6666666666666666666666666667f;
    }
    if (s05 >= s0 && s05 >= s_05)
    {
      max = s05;
      return 1.3333333333333333333333333333f;
    }
  }

  int two_b = -21 * i_05 + 36 * i0 - 15 * i05;
  // calculate max location:
  float ret_val = -float(two_b) / float(2 * two_a);
  // saturate and return
  if (ret_val < 0.6666666666666666666666666667f)
    ret_val = 0.666666666666666666666666667f;
  else if (ret_val > 1.33333333333333333333333333f)
    ret_val = 1.333333333333333333333333333f;
  int two_c = +12 * i_05 - 16 * i0 + 6 * i05;
  max = float(two_c) + float(two_a) * ret_val * ret_val + float(two_b) * ret_val;
  max /= 2048.0f;
  return ret_val;
}

//直接移植
__device__ inline float
BriskScaleSpace::refine1D_2(const float s_05, const float s0, const float s05, float& max) const
{
  int i_05 = int(1024.0 * s_05 + 0.5);
  int i0 = int(1024.0 * s0 + 0.5);
  int i05 = int(1024.0 * s05 + 0.5);

  //   18.0000  -30.0000   12.0000
  //  -45.0000   65.0000  -20.0000
  //   27.0000  -30.0000    8.0000

  int a = 2 * i_05 - 4 * i0 + 2 * i05;
  // second derivative must be negative:
  if (a >= 0)
  {
    if (s0 >= s_05 && s0 >= s05)
    {
      max = s0;
      return 1.0f;
    }
    if (s_05 >= s0 && s_05 >= s05)
    {
      max = s_05;
      return 0.7f;
    }
    if (s05 >= s0 && s05 >= s_05)
    {
      max = s05;
      return 1.5f;
    }
  }

  int b = -5 * i_05 + 8 * i0 - 3 * i05;
  // calculate max location:
  float ret_val = -float(b) / float(2 * a);
  // saturate and return
  if (ret_val < 0.7f)
    ret_val = 0.7f;
  else if (ret_val > 1.5f)
    ret_val = 1.5f; // allow to be slightly off bounds ...?
  int c = +3 * i_05 - 3 * i0 + 1 * i05;
  max = float(c) + float(a) * ret_val * ret_val + float(b) * ret_val;
  max /= 1024;
  return ret_val;
}


//直接移植
/***
 * 猜想：9个像素的方格不知道干啥
 * 反正返回的是一个9点插值亮度？
 * 何必呢
 * @param s_0_0
 * @param s_0_1
 * @param s_0_2
 * @param s_1_0
 * @param s_1_1
 * @param s_1_2
 * @param s_2_0
 * @param s_2_1
 * @param s_2_2
 * @param delta_x
 * @param delta_y
 * @return
 */
__device__ inline float
BriskScaleSpace::subpixel2D(const int s_0_0, const int s_0_1, const int s_0_2, const int s_1_0, const int s_1_1,
                            const int s_1_2, const int s_2_0, const int s_2_1, const int s_2_2, float& delta_x,
                            float& delta_y) const
{

  // the coefficients of the 2d quadratic function least-squares fit:
  int tmp1 = s_0_0 + s_0_2 - 2 * s_1_1 + s_2_0 + s_2_2;
  int coeff1 = 3 * (tmp1 + s_0_1 - ((s_1_0 + s_1_2) << 1) + s_2_1);
  int coeff2 = 3 * (tmp1 - ((s_0_1 + s_2_1) << 1) + s_1_0 + s_1_2);
  int tmp2 = s_0_2 - s_2_0;
  int tmp3 = (s_0_0 + tmp2 - s_2_2);
  int tmp4 = tmp3 - 2 * tmp2;
  int coeff3 = -3 * (tmp3 + s_0_1 - s_2_1);
  int coeff4 = -3 * (tmp4 + s_1_0 - s_1_2);
  int coeff5 = (s_0_0 - s_0_2 - s_2_0 + s_2_2) << 2;
  int coeff6 = -(s_0_0 + s_0_2 - ((s_1_0 + s_0_1 + s_1_2 + s_2_1) << 1) - 5 * s_1_1 + s_2_0 + s_2_2) << 1;

  // 2nd derivative test:
  int H_det = 4 * coeff1 * coeff2 - coeff5 * coeff5;

  if (H_det == 0)
  {
    delta_x = 0.0f;
    delta_y = 0.0f;
    return float(coeff6) / 18.0f;
  }

  if (!(H_det > 0 && coeff1 < 0))
  {
    // The maximum must be at the one of the 4 patch corners.
    int tmp_max = coeff3 + coeff4 + coeff5;
    delta_x = 1.0f;
    delta_y = 1.0f;

    int tmp = -coeff3 + coeff4 - coeff5;
    if (tmp > tmp_max)
    {
      tmp_max = tmp;
      delta_x = -1.0f;
      delta_y = 1.0f;
    }
    tmp = coeff3 - coeff4 - coeff5;
    if (tmp > tmp_max)
    {
      tmp_max = tmp;
      delta_x = 1.0f;
      delta_y = -1.0f;
    }
    tmp = -coeff3 - coeff4 + coeff5;
    if (tmp > tmp_max)
    {
      tmp_max = tmp;
      delta_x = -1.0f;
      delta_y = -1.0f;
    }
    return float(tmp_max + coeff1 + coeff2 + coeff6) / 18.0f;
  }

  // this is hopefully the normal outcome of the Hessian test
  delta_x = float(2 * coeff2 * coeff3 - coeff4 * coeff5) / float(-H_det);
  delta_y = float(2 * coeff1 * coeff4 - coeff3 * coeff5) / float(-H_det);
  // TODO: this is not correct, but easy, so perform a real boundary maximum search:
  bool tx = false;
  bool tx_ = false;
  bool ty = false;
  bool ty_ = false;
  if (delta_x > 1.0)
    tx = true;
  else if (delta_x < -1.0)
    tx_ = true;
  if (delta_y > 1.0)
    ty = true;
  if (delta_y < -1.0)
    ty_ = true;

  if (tx || tx_ || ty || ty_)
  {
    // get two candidates:
    float delta_x1 = 0.0f, delta_x2 = 0.0f, delta_y1 = 0.0f, delta_y2 = 0.0f;
    if (tx)
    {
      delta_x1 = 1.0f;
      delta_y1 = -float(coeff4 + coeff5) / float(2 * coeff2);
      if (delta_y1 > 1.0f)
        delta_y1 = 1.0f;
      else if (delta_y1 < -1.0f)
        delta_y1 = -1.0f;
    }
    else if (tx_)
    {
      delta_x1 = -1.0f;
      delta_y1 = -float(coeff4 - coeff5) / float(2 * coeff2);
      if (delta_y1 > 1.0f)
        delta_y1 = 1.0f;
      else if (delta_y1 < -1.0)
        delta_y1 = -1.0f;
    }
    if (ty)
    {
      delta_y2 = 1.0f;
      delta_x2 = -float(coeff3 + coeff5) / float(2 * coeff1);
      if (delta_x2 > 1.0f)
        delta_x2 = 1.0f;
      else if (delta_x2 < -1.0f)
        delta_x2 = -1.0f;
    }
    else if (ty_)
    {
      delta_y2 = -1.0f;
      delta_x2 = -float(coeff3 - coeff5) / float(2 * coeff1);
      if (delta_x2 > 1.0f)
        delta_x2 = 1.0f;
      else if (delta_x2 < -1.0f)
        delta_x2 = -1.0f;
    }
    // insert both options for evaluation which to pick
    float max1 = (coeff1 * delta_x1 * delta_x1 + coeff2 * delta_y1 * delta_y1 + coeff3 * delta_x1 + coeff4 * delta_y1
                  + coeff5 * delta_x1 * delta_y1 + coeff6)
                 / 18.0f;
    float max2 = (coeff1 * delta_x2 * delta_x2 + coeff2 * delta_y2 * delta_y2 + coeff3 * delta_x2 + coeff4 * delta_y2
                  + coeff5 * delta_x2 * delta_y2 + coeff6)
                 / 18.0f;
    if (max1 > max2)
    {
      delta_x = delta_x1;
      delta_y = delta_y1;
      return max1;
    }
    else
    {
      delta_x = delta_x2;
      delta_y = delta_y2;
      return max2;
    }
  }

  // this is the case of the maximum inside the boundaries:
  return (coeff1 * delta_x * delta_x + coeff2 * delta_y * delta_y + coeff3 * delta_x + coeff4 * delta_y
          + coeff5 * delta_x * delta_y + coeff6)
         / 18.0f;
}


