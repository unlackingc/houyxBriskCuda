#include "hip/hip_runtime.h"
/*
 * test.cpp
 *
 *  Created on: 2017年7月25日
 *      Author: houyx
 */


#include "FastCuda.h"

#include <npp.h>
#include <hip/hip_runtime.h>


using namespace std;
using namespace cv;
using namespace cv::cuda;

GpuMat createMat(Size size, int type, bool useRoi = false)
{
    Size size0 = size;

/*    if (useRoi)//todo: parse more
    {
        size0.width += randomInt(5, 15);
        size0.height += randomInt(5, 15);
    }*/

    GpuMat d_m(size0, type);

    if (size0 != size)
        d_m = d_m(Rect((size0.width - size.width) / 2, (size0.height - size.height) / 2, size.width, size.height));

    return d_m;
}

GpuMat loadMat(const Mat& m, bool useRoi = false)
{
    GpuMat d_m = createMat(m.size(), m.type(), useRoi);
    d_m.upload(m);
    return d_m;
}

#define idx(i,j) (j*cols + i)
void checkContentWithGpu( unsigned char* dcpu, unsigned char* dgpu, int rows, int cols)
{
	ofstream dout("debug.txt");
	unsigned char* temp;
	temp = new unsigned char[rows*cols];
	hipMemcpy(temp, dgpu, sizeof(unsigned char)*rows*cols, hipMemcpyDeviceToHost) ;

	int temp1,temp2;
	for( int i = 0; i < cols; i ++ )
	{
		for( int j = 0; j < rows; j ++)
		{
			temp1 = (unsigned char)(dcpu[idx(i,j)]), temp2 = (unsigned char)(temp[idx(i,j)]);
			//cout << hex << dcpu[idx(i,j)] << " ->G:-> " << temp[idx(i,j)] << endl;
			if( i%640 ==1)
			cout << i<<"-" << j <<":\t"<< temp1 << " ::: " << temp2 << endl;
			dout << i<<"-" << j <<":\t"<< temp1 << " ::: " << temp2 << endl;
		}
	}
	dout.close();
	free(temp);
}

void outputGpuMat( InputArray _image, unsigned char* dgpu, int rows, int cols )
{
    ofstream dout("debugcv2.txt");
    unsigned char* temp;
    temp = new unsigned char[rows*cols];
    hipMemcpy(temp, dgpu, sizeof(unsigned char)*rows*cols, hipMemcpyDeviceToHost) ;

    int temp1;

    for( int i = 0; i < cols; i ++ )
    {
        for( int j = 0; j < rows; j ++)
        {
            temp1 = (unsigned char)(temp[j*cols+i]);
            //cout << hex << dcpu[idx(i,j)] << " ->G:-> " << temp[idx(i,j)] << endl;
            cout << i<<"-" << j <<":\t"<< temp1 << endl;
            dout << i<<"-" << j <<":\t"<< temp1 << endl;
        }
    }
    dout.close();
    free(temp);
}

//todo: 传递data等必要数据，调用detect.
int main()
{
	cout << "hello world, I am in test.cu->main()" <<endl;
	int max_npoints_ = 2000;

	Mat testImg = imread( "data/test1.jpg" );
	Mat testImgGrayCpu;
	cv::cvtColor(testImg, testImgGrayCpu, CV_BGR2GRAY);
	if( !testImg.data )
	{
		cout <<"load data failed" <<endl;
	}

	imshow("test", testImgGrayCpu);
	//waitKey();

	//opencv origin
	Ptr<cv::cuda::FastFeatureDetector> fastDetector_;
	fastDetector_ = cuda::FastFeatureDetector::create();


	GpuMat fastKpRange;
	fastDetector_->detectAsync(loadMat(testImgGrayCpu), fastKpRange);
	//Stream& stream = Stream::Null();


	Mat keyPointsCpu(1, max_npoints_, CV_16SC2);
	Mat locCpu(1, max_npoints_, CV_16SC2);
	Mat responseCpu(1, max_npoints_, CV_32SC1);
	Mat scoreCpu(testImgGrayCpu.size(), CV_32SC1);

	GpuMat testImgGray = loadMat(testImgGrayCpu);
	GpuMat keyPoints = loadMat(keyPointsCpu);
	GpuMat loc = loadMat(locCpu);
	GpuMat response = loadMat(responseCpu);
	GpuMat score = loadMat(scoreCpu);
	score.setTo(Scalar::all(0));

	GpuMat _keypoints;
    ensureSizeIsEnough(cuda::FastFeatureDetector::ROWS_COUNT, 5000, CV_32FC1, _keypoints);
    //GpuMat& keypoints = _keypoints.getGpuMatRef();

	//score.
	//checkContentWithGpu((unsigned char*)(testImgGrayCpu.data), testImgGray.data, testImgGrayCpu.rows,testImgGrayCpu.cols);
	//outputGpuMat( testImgGrayCpu, loadMat(testImgGrayCpu).data, testImgGrayCpu.rows, testImgGrayCpu.cols );
	//outputGpuMat( testImgGrayCpu, testImgGray.data, testImgGrayCpu.rows, testImgGrayCpu.cols );
	//waitKey();
    //selfmade
	//int count = detectMe(testImgGray.rows, testImgGray.cols, testImgGray.step, testImgGray.data, keyPoints.ptr<short2>(), (int*)score.data, _keypoints.ptr<short2>(cuda::FastFeatureDetector::LOCATION_ROW), _keypoints.ptr<float>(cuda::FastFeatureDetector::RESPONSE_ROW));
	//detectMe(testImgGray.rows, testImgGray.cols, testImgGray.step, testImgGray.data, keyPoints.ptr<short2>(), (int*)score.data, loc.ptr<short2>(), (float*)response.data);

	int count = detectMe1( loadMat(testImgGrayCpu), _keypoints.ptr<short2>(cuda::FastFeatureDetector::LOCATION_ROW), score,  _keypoints.ptr<short2>(cuda::FastFeatureDetector::LOCATION_ROW), _keypoints.ptr<float>(cuda::FastFeatureDetector::RESPONSE_ROW) );


	//detectMe(int rows, int cols, unsigned char* image, short2* keyPoints, int* scores, short2* loc, float* response,int threshold=20, int maxPoints=2000, bool ifNoMaxSup = true);
	_keypoints.cols = count;
    int drawmode = DrawMatchesFlags::DRAW_RICH_KEYPOINTS;
    Mat result;
    vector<KeyPoint> KpRange;
    fastDetector_->convert(fastKpRange, KpRange);
    drawKeypoints(testImg, KpRange, result, Scalar::all(-1), drawmode);
	imshow("result_opencv_Gpu", result);
	//waitKey();
    cout <<"cols: " << testImgGray.cols << "\nrows: " << testImgGray.rows \
    		<< "\nelemSize: " << testImgGray.elemSize() << "\nstep: " << testImgGray.step <<endl;

    Mat result1;
    vector<KeyPoint> KpRangeGpu;
    cout << "there1" << endl;
    fastDetector_->convert(_keypoints, KpRangeGpu);
    cout << "there2" << endl;
    drawKeypoints(testImg, KpRangeGpu, result1, Scalar::all(-1), drawmode);

	imshow("result_Gpu", result1);
	waitKey();


    cout<<"size of description of Img: "<<fastKpRange.size()<<endl;
/*    for( int i = 0; i < KpRange.size(); i ++ )
    {
    	cout << "key point " <<i << ":\t" << KpRange[i].pt.x <<"\t" << KpRange[i].pt.y <<endl;
    }*/
	cout << "starting" << endl;



	//npp Start


	float nScaleFactor = 2.0/3.0;
	float shiftFactor = 0;

	NppiSize srcSize,dstSize;
	srcSize.height = testImg.rows;
	srcSize.width = testImg.cols;


	//unsigned char * dstImage;

	//hipMalloc(&dstImage, dstSize.height * dstSize.width );

	NppiInterpolationMode eInterploationMode = NPPI_INTER_SUPER;


	NppiRect oSrcImageROI = {0,0,srcSize.width, srcSize.height};
	NppiRect oDstImageROI;

	nppiGetResizeRect(oSrcImageROI, &oDstImageROI,
	                                        nScaleFactor,
	                                        nScaleFactor,
	                                        shiftFactor, shiftFactor, eInterploationMode);


	dstSize.height = oDstImageROI.height ;//+ (srcSize.height%3==0)?0:1;
	dstSize.width = oDstImageROI.width ;//+ (srcSize.width%3==0)?0:1;

	GpuMat dstImage1;
	//ensureSizeIsEnough(cuda::FastFeatureDetector::ROWS_COUNT, 5000, CV_32FC1, _keypoints);
	//ensureSizeIsEnough(dstSize.height, dstSize.width, CV_8UC1, dstImage1);
	unsigned char * dstImagedata;

	hipMalloc(&dstImagedata, dstSize.height * dstSize.width );
	dstImage1.data = dstImagedata;
	dstImage1.cols = dstSize.width;
	dstImage1.step = dstSize.width;
	dstImage1.rows = dstSize.height;

	nppiResizeSqrPixel_8u_C1R(testImgGray.data, srcSize, testImgGray.step, oSrcImageROI,
			dstImage1.data, dstImage1.step, oDstImageROI,
	        nScaleFactor,
	        nScaleFactor,
	        shiftFactor, shiftFactor, eInterploationMode);


	GpuMat integralData;
	//ensureSizeIsEnough(cuda::FastFeatureDetector::ROWS_COUNT, 5000, CV_32FC1, _keypoints);
	ensureSizeIsEnough(dstSize.height + 1, dstSize.width + 1, CV_32SC1, integralData);
	nppiIntegral_8u32s_C1R (dstImage1.data, dstImage1.step, (Npp32s*)(integralData.data), integralData.step, dstSize, 0 );
	//nppiIntegral_8u32s_C1R (const Npp8u *pSrc, int nSrcStep, Npp32s *pDst, int nDstStep, NppiSize oROI, Npp32s nVal)

	Mat resizedCpu(dstSize.height, dstSize.width, CV_8UC1);
	Mat resizedCpu1(dstSize.height, dstSize.width, CV_8UC1);
	Mat resizedCpu2(dstSize.height+1, dstSize.width+1, CV_8UC1);
	testImgGray.download(resizedCpu);
	dstImage1.download(resizedCpu1);
	integralData.download(resizedCpu2);
	//hipMemcpy(resizedCpu.data,dstImage,dstSize.width*dstSize.height,hipMemcpyDeviceToHost);

	printf("haha I'm here");

	imshow("resize_result", resizedCpu);
	imshow("resize_result11111", resizedCpu1);
	imshow("resize_result22", resizedCpu2);
		waitKey();

	return 0;
}


