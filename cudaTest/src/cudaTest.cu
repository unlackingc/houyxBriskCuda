#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : cudaTest.cu
 Author      : houyx
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include "cuda_types.hpp"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "memory.h"
#include "string.h"

using namespace std;


const int size = 15;
static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}


class VV1
{
public:
	int val;
	int myarray[size];
	int* testfuck;
	int count;


	VV1();

	~VV1()
	{
		cout << "in ~VV1" << endl;
		//if( testfuck != NULL )
		if( count == 0 )
			hipFree(testfuck);
	}

	VV1(const VV1& c)
	{
		cout << "in VV1 copy" << endl;
		//val = c.val;
		*this=c;
		count = c.count+1;
	}

};

class VV
{
public:


	int val;
	int myarray[size];
	int* testfuck;
	int count;
	VV1 vv1;

	VV();

	~VV()
	{
		cout << "in ~VV" << endl;
		//if( testfuck != NULL )
		if( count == 0 )
			hipFree(testfuck);
	}

	VV(const VV& c):vv1(c.vv1)
	{
		cout << "in VV copy" << endl;
		//val = c.val;
		*this=c;
		count = c.count+1;
		vv1.count = c.vv1.count + 1;
	}

	__device__ void changeSelf()
	{
		val = val + 3;
	}


	void testGlobal();
};

__global__ void setVal( int* testfuck, int size )
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	testfuck[id] = size - id;
}


VV::VV():val(11),count(0)
	{
	cout << "in VV" << endl;
		hipMalloc((void**)&testfuck, sizeof(int)*size);

		//cout << "what's the fuck3" << endl;
		//myarray = new int[size];
		for( int i = 0; i < size; i++ )
		{
			myarray[i] = i*2;
			//testr[i] = size - i;
		}
		setVal<<<1,size>>>(testfuck,size);
		//cout << "what's the fuck4" << endl;
	}

VV1::VV1():val(1),count(0)
	{
	cout << "in VV1" << endl;
		hipMalloc((void**)&testfuck, sizeof(int)*size);

		//cout << "what's the fuck6" << endl;
		//myarray = new int[size];
		for( int i = 0; i < size; i++ )
		{
			myarray[i] = i*2;
			//testr[i] = size - i;
		}
		setVal<<<1,size>>>(testfuck,size);
	}


__global__ void changeVal( VV* a )
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	a->myarray[id] =a->myarray[id]*2;
	a->val = 2*a->val;
	a->changeSelf();
	//a->testr[id] = a->testr[id] - 2*a->testr[id];
}


__global__ void changeVal1( VV a )
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	a.myarray[id] =a.myarray[id]*2;
	a.val = 2*a.val;
	a.changeSelf();
	//a->testr[id] = a->testr[id] - 2*a->testr[id];
}

__global__ void getVal( VV a, int* ret )
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	//a.myarray[id] *=2;
	//ret[id] = a.testr[id];
}


void VV::testGlobal()
{
	VV* aInner;
	hipMalloc((void**)&aInner, sizeof(VV));
	hipMemcpy(aInner,this,sizeof(VV),hipMemcpyHostToDevice);

	changeVal<<<1,size>>>(aInner);

	hipMemcpy(this,aInner,sizeof(VV),hipMemcpyDeviceToHost);


	int * tempInner;;

	hipMalloc((void**)&tempInner, sizeof(int) * size);
	hipMemset ( tempInner, 0, sizeof(int)*size );



	getVal<<<1,size>>>(*this,tempInner);

	int tempOuter[size];

	memset( tempOuter, 0, size*sizeof(int) );

	hipMemcpy(tempOuter,tempInner,sizeof(int)*size,hipMemcpyDeviceToHost);


	for( int i = 0; i < size; i ++ )
	{
		cout << "after::\t" << i << "::: " << tempOuter[i] << endl;
	}


	//hipFree(tempInner);
}

__global__ void kernel(VV a)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	a.testfuck[id] = a.myarray[id];
	a.vv1.testfuck[id] = id*10;
}


int main(void)
{

	VV a;



	int tempOuter[size];

	memset( tempOuter, 0, size*sizeof(int) );

	hipMemcpy(tempOuter,a.vv1.testfuck,sizeof(int)*size,hipMemcpyDeviceToHost);


	for( int i = 0; i < size; i ++ )
	{
		cout << "after::\t" << i << "::: " << tempOuter[i] << endl;
	}


	kernel<<<1,size>>>(a);

	memset( tempOuter, 0, size*sizeof(int) );

	hipMemcpy(tempOuter,a.vv1.testfuck,sizeof(int)*size,hipMemcpyDeviceToHost);


	for( int i = 0; i < size; i ++ )
	{
		cout << "after1::\t" << i << "::: " << tempOuter[i] << endl;
	}

	return 0;
}

