#include "hip/hip_runtime.h"
/*
 * test.cpp
 *
 *  Created on: 2017年7月25日
 *      Author: houyx
 */

#include "briskCode/BriskScaleSpace.cuh"

#include "opencv2/cudafeatures2d.hpp"
#include <opencv2/features2d/features2d.hpp>
#include <opencv/cv.h>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/core/core.hpp>

#include <vector>

using namespace std;

void poutfloat2(float2* m, int size, std::string info) {
	float2 temp;
	memset(&temp, 0, sizeof(float2));
	std::cout << info << std::endl;
	for (int i = 0; i < size; i++) {
		CUDA_CHECK_RETURN(
				hipMemcpy(&temp, &m[i], sizeof(float2),
						hipMemcpyDeviceToHost));
		std::cout << "====" << info << "==== "<<i<<": (" << temp.x << "," << temp.y
				<< ")";
		std::cout << std::endl;
	}
	std::cout << "******************finish*******************" << std::endl;
}

void copyToKeyPoint(vector<cv::KeyPoint>& keypoints1, int size,
		float2* keypoints, float* kpSize, float* kpScore) {

	keypoints1.clear();
	float2 kptemp;
	float kpsizetemp;
	float kpscoretemp;

	for( int i = 0; i < size; i ++ )
	{
		CUDA_CHECK_RETURN(hipMemcpy(&kptemp, &keypoints[i], sizeof(float2), hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpy(&kpsizetemp, &kpSize[i], sizeof(float), hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpy(&kpscoretemp, &kpScore[i], sizeof(float), hipMemcpyDeviceToHost));
		keypoints1.push_back(cv::KeyPoint(float(kptemp.x), float(kptemp.y), kpsizetemp, -1, kpscoretemp, 0));
	}
}

void copyDescritpor( PtrStepSzb desGpu, cv::Mat& descriptor, int size, int singleSize )
{
	descriptor.create(size,singleSize,CV_8U);



	for( int i = 0; i < size; i++ )
	{
		CUDA_CHECK_RETURN(hipMemcpy(descriptor.ptr<unsigned char>(i), &(desGpu.data[i*singleSize]), sizeof(unsigned char)*singleSize, hipMemcpyDeviceToHost));
	}
	//descritpor.create((size, singleSize, CV_8U);
}


//todo:把BirskScaleSpace放入构造函数，改进数组分配，避免多次检测时不必要的数据分配


int main() {
	cv::Mat testImg = cv::imread("data/test1.jpg");

	cv::Mat testResize;
	testResize.create(testImg.rows / 2, testImg.cols / 2, CV_8U);
	cv::resize(testImg,testResize,testResize.size(),0,0,cv::INTER_AREA);

	cv::Mat testRotate;
	cv::transpose(testImg,testRotate);

	cv::Mat testImgGray;
	cv::cvtColor(testRotate, testImgGray, CV_BGR2GRAY);
	if (!testImg.data) {
		cout << "load data failed" << endl;
	}
	//cv::imshow("test", testImgGray);
	//cv::waitKey();

	cv::cuda::GpuMat dstImage1;
	//ensureSizeIsEnough(cuda::FastFeatureDetector::ROWS_COUNT, 5000, CV_32FC1, _keypoints);
	//ensureSizeIsEnough(dstSize.height, dstSize.width, CV_8UC1, dstImage1);
	unsigned char * dstImagedata;

	hipMalloc(&dstImagedata, testImgGray.rows * testImgGray.cols);

	for (int i = 0; i < testImgGray.rows; i++) {
		hipMemcpy(dstImagedata + i * testImgGray.cols,
				testImgGray.data + i * testImgGray.step,
				sizeof(unsigned char) * testImgGray.cols,
				hipMemcpyHostToDevice);
	}

	dstImage1.data = dstImagedata;
	dstImage1.cols = testImgGray.cols;
	dstImage1.step = testImgGray.cols;
	dstImage1.rows = testImgGray.rows;

	cv::Mat retestCpu(testImgGray.rows, testImgGray.cols, CV_8UC1);
	dstImage1.download(retestCpu);

	cv::imshow("retestCpu", retestCpu);
	cv::waitKey();
//(int rows_, int cols_, T* data_, size_t step_)
	PtrStepSzb imageIn(dstImage1.rows, dstImage1.cols, dstImage1.data,
			dstImage1.step);


	cout << "load image done!!" << endl;

	BRISK_Impl a(dstImage1.rows, dstImage1.cols);
	int2 size = a.detectAndCompute(imageIn, a.keypointsG, a.kpSizeG, a.kpScoreG,
			false);


	cv::Mat descriptors;
	copyDescritpor( a.descriptorsG, descriptors, size.y, a.strings_ );







	cout << size.x << " " << size.y << endl;
	poutfloat2(a.keypointsG, size.x, "keypointsG");
	pouta(a.kpSizeG, size.x, "kpSizeG");
	pouta(a.kpScoreG, size.x, "kpScoreG");

	//display
	vector<cv::KeyPoint> keypoints;
	copyToKeyPoint(keypoints, size.x, a.keypointsG, a.kpSizeG, a.kpScoreG);

	cv::Mat result1;
	int drawmode = cv::DrawMatchesFlags::DRAW_RICH_KEYPOINTS;
	//fastDetector_->convert(fastKpRange, KpRange);
	cv::drawKeypoints(testImgGray, keypoints, result1, cv::Scalar::all(-1),
			drawmode);

	cv::imshow("result1", result1);
	cv::waitKey();

	cout << "end!!" << endl;


	cout << "des size: " << a.strings_ << endl;
	return 0;
}
